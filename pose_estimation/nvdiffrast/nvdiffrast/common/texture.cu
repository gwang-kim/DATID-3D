#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include "common.h"
#include "texture.h"

//------------------------------------------------------------------------
// Memory access and math helpers.

static __device__ __forceinline__ void accum_from_mem(float* a, int s, float  b, float c) { a[0] += b * c; }
static __device__ __forceinline__ void accum_from_mem(float* a, int s, float2 b, float c) { a[0] += b.x * c; a[s] += b.y * c; }
static __device__ __forceinline__ void accum_from_mem(float* a, int s, float4 b, float c) { a[0] += b.x * c; a[s] += b.y * c; a[2*s] += b.z * c; a[3*s] += b.w * c; }
static __device__ __forceinline__ void accum_to_mem(float&  a, float* b, int s) { a += b[0]; }
static __device__ __forceinline__ void accum_to_mem(float2& a, float* b, int s) { float2 v = a; v.x += b[0]; v.y += b[s]; a = v; }
static __device__ __forceinline__ void accum_to_mem(float4& a, float* b, int s) { float4 v = a; v.x += b[0]; v.y += b[s]; v.z += b[2*s]; v.w += b[3*s]; a = v; }
template<class T> static __device__ __forceinline__ T lerp  (const T& a, const T& b, float c) { return a + c * (b - a); }
template<class T> static __device__ __forceinline__ T bilerp(const T& a, const T& b, const T& c, const T& d, const float2& e) { return lerp(lerp(a, b, e.x), lerp(c, d, e.x), e.y); }

//------------------------------------------------------------------------
// Cube map wrapping for smooth filtering across edges and corners. At corners,
// one of the texture coordinates will be negative. For correct interpolation,
// the missing texel must take the average color of the other three.

static __constant__ uint32_t c_cubeWrapMask1[48] =
{
    0x1530a440, 0x1133a550, 0x6103a110, 0x1515aa44, 0x6161aa11, 0x40154a04, 0x44115a05, 0x04611a01,
    0x2630a440, 0x2233a550, 0x5203a110, 0x2626aa44, 0x5252aa11, 0x40264a04, 0x44225a05, 0x04521a01,
    0x32608064, 0x3366a055, 0x13062091, 0x32328866, 0x13132299, 0x50320846, 0x55330a55, 0x05130219,
    0x42508064, 0x4455a055, 0x14052091, 0x42428866, 0x14142299, 0x60420846, 0x66440a55, 0x06140219,
    0x5230a044, 0x5533a055, 0x1503a011, 0x5252aa44, 0x1515aa11, 0x40520a44, 0x44550a55, 0x04150a11,
    0x6130a044, 0x6633a055, 0x2603a011, 0x6161aa44, 0x2626aa11, 0x40610a44, 0x44660a55, 0x04260a11,
};

static __constant__ uint8_t c_cubeWrapMask2[48] =
{
    0x26, 0x33, 0x11, 0x05, 0x00, 0x09, 0x0c, 0x04, 0x04, 0x00, 0x00, 0x05, 0x00, 0x81, 0xc0, 0x40,
    0x02, 0x03, 0x09, 0x00, 0x0a, 0x00, 0x00, 0x02, 0x64, 0x30, 0x90, 0x55, 0xa0, 0x99, 0xcc, 0x64,
    0x24, 0x30, 0x10, 0x05, 0x00, 0x01, 0x00, 0x00, 0x06, 0x03, 0x01, 0x05, 0x00, 0x89, 0xcc, 0x44,
};

static __device__ __forceinline__ int4 wrapCubeMap(int face, int ix0, int ix1, int iy0, int iy1, int w)
{
    // Calculate case number.
    int cx = (ix0 < 0) ? 0 : (ix1 >= w) ? 2 : 1;
    int cy = (iy0 < 0) ? 0 : (iy1 >= w) ? 6 : 3;
    int c = cx + cy;
    if (c >= 5)
        c--;
    c = (face << 3) + c;

    // Compute coordinates and faces.
    unsigned int m = c_cubeWrapMask1[c];
    int x0 = (m >>  0) & 3; x0 = (x0 == 0) ? 0 : (x0 == 1) ? ix0 : iy0;
    int x1 = (m >>  2) & 3; x1 = (x1 == 0) ? 0 : (x1 == 1) ? ix1 : iy0;
    int x2 = (m >>  4) & 3; x2 = (x2 == 0) ? 0 : (x2 == 1) ? ix0 : iy1;
    int x3 = (m >>  6) & 3; x3 = (x3 == 0) ? 0 : (x3 == 1) ? ix1 : iy1;
    int y0 = (m >>  8) & 3; y0 = (y0 == 0) ? 0 : (y0 == 1) ? ix0 : iy0;
    int y1 = (m >> 10) & 3; y1 = (y1 == 0) ? 0 : (y1 == 1) ? ix1 : iy0;
    int y2 = (m >> 12) & 3; y2 = (y2 == 0) ? 0 : (y2 == 1) ? ix0 : iy1;
    int y3 = (m >> 14) & 3; y3 = (y3 == 0) ? 0 : (y3 == 1) ? ix1 : iy1;
    int f0 = ((m >> 16) & 15) - 1;
    int f1 = ((m >> 20) & 15) - 1;
    int f2 = ((m >> 24) & 15) - 1;
    int f3 = ((m >> 28)     ) - 1;

    // Flips.
    unsigned int f = c_cubeWrapMask2[c];
    int w1 = w - 1;
    if (f & 0x01) x0 = w1 - x0;
    if (f & 0x02) x1 = w1 - x1;
    if (f & 0x04) x2 = w1 - x2;
    if (f & 0x08) x3 = w1 - x3;
    if (f & 0x10) y0 = w1 - y0;
    if (f & 0x20) y1 = w1 - y1;
    if (f & 0x40) y2 = w1 - y2;
    if (f & 0x80) y3 = w1 - y3;

    // Done.
    int4 tcOut;
    tcOut.x = x0 + (y0 + f0 * w) * w;
    tcOut.y = x1 + (y1 + f1 * w) * w;
    tcOut.z = x2 + (y2 + f2 * w) * w;
    tcOut.w = x3 + (y3 + f3 * w) * w;
    return tcOut;
}

//------------------------------------------------------------------------
// Cube map indexing and gradient functions.

// Map a 3D lookup vector into an (s,t) face coordinates (returned in first .
// two parameters) and face index.
static __device__ __forceinline__ int indexCubeMap(float& x, float& y, float z)
{
    float ax = fabsf(x);
    float ay = fabsf(y);
    float az = fabsf(z);
    int idx;
    float c;
    if (az > fmaxf(ax, ay)) { idx = 4; c = z; }
    else if (ay > ax)       { idx = 2; c = y; y = z; }
    else                    { idx = 0; c = x; x = z; }
    if (c < 0.f) idx += 1;
    float m = __frcp_rz(fabsf(c)) * .5;
    float m0 = __uint_as_float(__float_as_uint(m) ^ ((0x21u >> idx) << 31));
    float m1 = (idx != 2) ? -m : m;
    x = x * m0 + .5;
    y = y * m1 + .5;
    x = fminf(fmaxf(x, 0.f), 1.f);
    y = fminf(fmaxf(y, 0.f), 1.f);
    return idx;
}

// Based on dA/d{s,t}, compute dA/d{x,y,z} at a given 3D lookup vector.
static __device__ __forceinline__ float3 indexCubeMapGrad(float3 uv, float gu, float gv)
{
    float ax = fabsf(uv.x);
    float ay = fabsf(uv.y);
    float az = fabsf(uv.z);
    int idx;
    float c;
    float c0 = gu;
    float c1 = gv;
    if (az > fmaxf(ax, ay)) { idx = 0x10; c = uv.z; c0 *= uv.x; c1 *= uv.y; }
    else if (ay > ax)       { idx = 0x04; c = uv.y; c0 *= uv.x; c1 *= uv.z; }
    else                    { idx = 0x01; c = uv.x; c0 *= uv.z; c1 *= uv.y; }
    if (c < 0.f) idx += idx;
    float m = __frcp_rz(fabsf(c));
    c0 = (idx & 0x34) ? -c0 : c0;
    c1 = (idx & 0x2e) ? -c1 : c1;
    float gl = (c0 + c1) * m;
    float gx = (idx & 0x03) ? gl : (idx & 0x20) ? -gu : gu;
    float gy = (idx & 0x0c) ? gl : -gv;
    float gz = (idx & 0x30) ? gl : (idx & 0x03) ? gu : gv;
    gz = (idx & 0x09) ? -gz : gz;
    return make_float3(gx, gy, gz) * (m * .5f);
}

// Based on dL/d(d{s,t}/s{X,Y}), compute dL/d(d{x,y,z}/d{X,Y}). This is just two
// indexCubeMapGrad() functions rolled together.
static __device__ __forceinline__ void indexCubeMapGrad4(float3 uv, float4 dw, float3& g0, float3& g1)
{
    float ax = fabsf(uv.x);
    float ay = fabsf(uv.y);
    float az = fabsf(uv.z);
    int idx;
    float c, c0, c1;
    if (az > fmaxf(ax, ay)) { idx = 0x10; c = uv.z; c0 = uv.x; c1 = uv.y; }
    else if (ay > ax)       { idx = 0x04; c = uv.y; c0 = uv.x; c1 = uv.z; }
    else                    { idx = 0x01; c = uv.x; c0 = uv.z; c1 = uv.y; }
    if (c < 0.f) idx += idx;
    float m = __frcp_rz(fabsf(c));
    c0 = (idx & 0x34) ? -c0 : c0;
    c1 = (idx & 0x2e) ? -c1 : c1;
    float gl0 = (dw.x * c0 + dw.z * c1) * m;
    float gl1 = (dw.y * c0 + dw.w * c1) * m;
    float gx0 = (idx & 0x03) ? gl0 : (idx & 0x20) ? -dw.x : dw.x;
    float gx1 = (idx & 0x03) ? gl1 : (idx & 0x20) ? -dw.y : dw.y;
    float gy0 = (idx & 0x0c) ? gl0 : -dw.z;
    float gy1 = (idx & 0x0c) ? gl1 : -dw.w;
    float gz0 = (idx & 0x30) ? gl0 : (idx & 0x03) ? dw.x : dw.z;
    float gz1 = (idx & 0x30) ? gl1 : (idx & 0x03) ? dw.y : dw.w;
    if (idx & 0x09)
    {
        gz0 = -gz0;
        gz1 = -gz1;
    }
    g0 = make_float3(gx0, gy0, gz0) * (m * .5f);
    g1 = make_float3(gx1, gy1, gz1) * (m * .5f);
}

// Compute d{s,t}/d{X,Y} based on d{x,y,z}/d{X,Y} at a given 3D lookup vector.
// Result is (ds/dX, ds/dY, dt/dX, dt/dY).
static __device__ __forceinline__ float4 indexCubeMapGradST(float3 uv, float3 dvdX, float3 dvdY)
{
    float ax = fabsf(uv.x);
    float ay = fabsf(uv.y);
    float az = fabsf(uv.z);
    int idx;
    float c, gu, gv;
    if (az > fmaxf(ax, ay)) { idx = 0x10; c = uv.z; gu = uv.x; gv = uv.y; }
    else if (ay > ax)       { idx = 0x04; c = uv.y; gu = uv.x; gv = uv.z; }
    else                    { idx = 0x01; c = uv.x; gu = uv.z; gv = uv.y; }
    if (c < 0.f) idx += idx;
    if (idx & 0x09)
    {
        dvdX.z = -dvdX.z;
        dvdY.z = -dvdY.z;
    }
    float m = __frcp_rz(fabsf(c));
    float dm = m * .5f;
    float mm = m * dm;
    gu *= (idx & 0x34) ? -mm : mm;
    gv *= (idx & 0x2e) ? -mm : mm;

    if (idx & 0x03)
    {
        return make_float4(gu * dvdX.x + dm * dvdX.z,
                           gu * dvdY.x + dm * dvdY.z,
                           gv * dvdX.x - dm * dvdX.y,
                           gv * dvdY.x - dm * dvdY.y);
    }
    else if (idx & 0x0c)
    {
        return make_float4(gu * dvdX.y + dm * dvdX.x,
                           gu * dvdY.y + dm * dvdY.x,
                           gv * dvdX.y + dm * dvdX.z,
                           gv * dvdY.y + dm * dvdY.z);
    } 
    else // (idx & 0x30)
    {
        return make_float4(gu * dvdX.z + copysignf(dm, c) * dvdX.x,
                           gu * dvdY.z + copysignf(dm, c) * dvdY.x,
                           gv * dvdX.z - dm * dvdX.y,
                           gv * dvdY.z - dm * dvdY.y);
    }
}

// Compute d(d{s,t}/d{X,Y})/d{x,y,z}, i.e., how the pixel derivatives of 2D face
// coordinates change w.r.t. 3D texture coordinate vector, returned as follows:
//   |  d(ds/dX)/dx  d(ds/dY)/dx  d(dt/dX)/dx  d(dt/dY)/dx  |
//   |  d(ds/dX)/dy  d(ds/dY)/dy  d(dt/dX)/dy  d(dt/dY)/dy  |
//   |  d(ds/dX)/dz  d(ds/dY)/dz  d(dt/dX)/dz  d(dt/dY)/dz  |
static __device__ __forceinline__ void indexCubeMapGrad2(float3 uv, float3 dvdX, float3 dvdY, float4& dx, float4& dy, float4& dz)
{
    float ax = fabsf(uv.x);
    float ay = fabsf(uv.y);
    float az = fabsf(uv.z);
    int idx;
    float c, gu, gv;
    if (az > fmaxf(ax, ay)) { idx = 0x10; c = uv.z; gu = uv.x; gv = uv.y; }
    else if (ay > ax)       { idx = 0x04; c = uv.y; gu = uv.x; gv = uv.z; }
    else                    { idx = 0x01; c = uv.x; gu = uv.z; gv = uv.y; }
    if (c < 0.f) idx += idx;

    if (idx & 0x09)
    {
        dvdX.z = -dvdX.z;
        dvdY.z = -dvdY.z;
    }

    float m = __frcp_rz(c);
    float dm = -m * fabsf(m) * .5;
    float mm = m * m * .5;
    float mu = (idx & 0x34) ? -mm : mm;
    float mv = (idx & 0x2e) ? -mm : mm;
    gu *= -2.0 * m * mu;
    gv *= -2.0 * m * mv;

    if (idx & 0x03)
    {
        dx.x = gu * dvdX.x + dm * dvdX.z;
        dx.y = gu * dvdY.x + dm * dvdY.z;
        dx.z = gv * dvdX.x - dm * dvdX.y;
        dx.w = gv * dvdY.x - dm * dvdY.y;
        dy.x = 0.f;
        dy.y = 0.f;
        dy.z = mv * dvdX.x;
        dy.w = mv * dvdY.x;
        dz.x = mu * dvdX.x;
        dz.y = mu * dvdY.x;
        dz.z = 0.f;
        dz.w = 0.f;
    }
    else if (idx & 0x0c)
    {
        dx.x = mu * dvdX.y;
        dx.y = mu * dvdY.y;
        dx.z = 0.f;
        dx.w = 0.f;
        dy.x = gu * dvdX.y + dm * dvdX.x;
        dy.y = gu * dvdY.y + dm * dvdY.x;
        dy.z = gv * dvdX.y + dm * dvdX.z;
        dy.w = gv * dvdY.y + dm * dvdY.z;
        dz.x = 0.f;
        dz.y = 0.f;
        dz.z = mv * dvdX.y;
        dz.w = mv * dvdY.y;
    }
    else // (idx & 0x30)
    {
        dx.x = mu * dvdX.z;
        dx.y = mu * dvdY.z;
        dx.z = 0.f;
        dx.w = 0.f;
        dy.x = 0.f;
        dy.y = 0.f;
        dy.z = mv * dvdX.z;
        dy.w = mv * dvdY.z;
        dz.x = gu * dvdX.z - fabsf(dm) * dvdX.x;
        dz.y = gu * dvdY.z - fabsf(dm) * dvdY.x;
        dz.z = gv * dvdX.z - dm * dvdX.y;
        dz.w = gv * dvdY.z - dm * dvdY.y;
    }
}

//------------------------------------------------------------------------
// General texture indexing.

template <bool CUBE_MODE>
static __device__ __forceinline__ int indexTextureNearest(const TextureKernelParams& p, float3 uv, int tz)
{
    int w = p.texWidth;
    int h = p.texHeight;
    float u = uv.x;
    float v = uv.y;

    // Cube map indexing.
    if (CUBE_MODE)
    {
        // No wrap. Fold face index into tz right away.
        tz = 6 * tz + indexCubeMap(u, v, uv.z); // Rewrites u, v.
    }
    else
    {
        // Handle boundary.
        if (p.boundaryMode == TEX_BOUNDARY_MODE_WRAP)
        {
            u = u - (float)__float2int_rd(u);
            v = v - (float)__float2int_rd(v);
        }
    }

    u = u * (float)w;
    v = v * (float)h;

    int iu = __float2int_rd(u);
    int iv = __float2int_rd(v);

    // In zero boundary mode, return texture address -1.
    if (!CUBE_MODE && p.boundaryMode == TEX_BOUNDARY_MODE_ZERO)
    {
        if (iu < 0 || iu >= w || iv < 0 || iv >= h)
            return -1;
    }

    // Otherwise clamp and calculate the coordinate properly.
    iu = min(max(iu, 0), w-1);
    iv = min(max(iv, 0), h-1);
    return iu + w * (iv + tz * h);
}

template <bool CUBE_MODE>
static __device__ __forceinline__ float2 indexTextureLinear(const TextureKernelParams& p, float3 uv, int tz, int4& tcOut, int level)
{
    // Mip level size.
    int2 sz = mipLevelSize(p, level);
    int w = sz.x;
    int h = sz.y;

    // Compute texture-space u, v.
    float u = uv.x;
    float v = uv.y;
    bool clampU = false;
    bool clampV = false;

    // Cube map indexing.
    int face = 0;
    if (CUBE_MODE)
    {
        // Neither clamp or wrap.
        face = indexCubeMap(u, v, uv.z); // Rewrites u, v.
        u = u * (float)w - 0.5f;
        v = v * (float)h - 0.5f;
    }
    else
    {
        if (p.boundaryMode == TEX_BOUNDARY_MODE_WRAP)
        {
            // Wrap.
            u = u - (float)__float2int_rd(u);
            v = v - (float)__float2int_rd(v);
        }

        // Move to texel space.
        u = u * (float)w - 0.5f;
        v = v * (float)h - 0.5f;

        if (p.boundaryMode == TEX_BOUNDARY_MODE_CLAMP)
        {
            // Clamp to center of edge texels.
            u = fminf(fmaxf(u, 0.f), w - 1.f);
            v = fminf(fmaxf(v, 0.f), h - 1.f);
            clampU = (u == 0.f || u == w - 1.f);
            clampV = (v == 0.f || v == h - 1.f);
        }
    }

    // Compute texel coordinates and weights.
    int iu0 = __float2int_rd(u);
    int iv0 = __float2int_rd(v);
    int iu1 = iu0 + (clampU ? 0 : 1); // Ensure zero u/v gradients with clamped.
    int iv1 = iv0 + (clampV ? 0 : 1);
    u -= (float)iu0;
    v -= (float)iv0;

    // Cube map wrapping.
    bool cubeWrap = CUBE_MODE && (iu0 < 0 || iv0 < 0 || iu1 >= w || iv1 >= h);
    if (cubeWrap)
    {
        tcOut = wrapCubeMap(face, iu0, iu1, iv0, iv1, w);
        tcOut += 6 * tz * w * h;  // Bring in tz.
        return make_float2(u, v); // Done.
    }

    // Fold cube map face into tz.
    if (CUBE_MODE)
        tz = 6 * tz + face;

    // Wrap overflowing texel indices.
    if (!CUBE_MODE && p.boundaryMode == TEX_BOUNDARY_MODE_WRAP)
    {
        if (iu0 < 0) iu0 += w;
        if (iv0 < 0) iv0 += h;
        if (iu1 >= w) iu1 -= w;
        if (iv1 >= h) iv1 -= h;
    }

    // Coordinates
    iu0 += tz * w * h;
    iu1 += tz * w * h;
    tcOut.x = iu0 + w * iv0;
    tcOut.y = iu1 + w * iv0;
    tcOut.z = iu0 + w * iv1;
    tcOut.w = iu1 + w * iv1;

    // Invalidate texture addresses outside unit square if we are in zero mode.
    if (!CUBE_MODE && p.boundaryMode == TEX_BOUNDARY_MODE_ZERO)
    {
        bool iu0_out = (iu0 < 0 || iu0 >= w);
        bool iu1_out = (iu1 < 0 || iu1 >= w);
        bool iv0_out = (iv0 < 0 || iv0 >= h);
        bool iv1_out = (iv1 < 0 || iv1 >= h);
        if (iu0_out || iv0_out) tcOut.x = -1;
        if (iu1_out || iv0_out) tcOut.y = -1;
        if (iu0_out || iv1_out) tcOut.z = -1;
        if (iu1_out || iv1_out) tcOut.w = -1;
    }

    // All done.
    return make_float2(u, v);
}

//------------------------------------------------------------------------
// Mip level calculation.

template <bool CUBE_MODE, bool BIAS_ONLY, int FILTER_MODE>
static __device__ __forceinline__ void calculateMipLevel(int& level0, int& level1, float& flevel, const TextureKernelParams& p, int pidx, float3 uv, float4* pdw, float3* pdfdv)
{
    // Do nothing if mips not in use.
    if (FILTER_MODE == TEX_MODE_NEAREST || FILTER_MODE == TEX_MODE_LINEAR)
        return;

    // Determine mip level based on UV pixel derivatives. If no derivatives are given (mip level bias only), leave as zero.
    if (!BIAS_ONLY)
    {
        // Get pixel derivatives of texture coordinates.
        float4 uvDA;
        float3 dvdX, dvdY; // Gradients use these later.
        if (CUBE_MODE)
        {
            // Fetch.
            float2 d0 = ((const float2*)p.uvDA)[3 * pidx + 0];
            float2 d1 = ((const float2*)p.uvDA)[3 * pidx + 1];
            float2 d2 = ((const float2*)p.uvDA)[3 * pidx + 2];

            // Map d{x,y,z}/d{X,Y} into d{s,t}/d{X,Y}.
            dvdX = make_float3(d0.x, d1.x, d2.x); // d{x,y,z}/dX
            dvdY = make_float3(d0.y, d1.y, d2.y); // d{x,y,z}/dY
            uvDA = indexCubeMapGradST(uv, dvdX, dvdY); // d{s,t}/d{X,Y}
        }
        else
        {
            // Fetch.
            uvDA = ((const float4*)p.uvDA)[pidx];
        }

        // Scaling factors.
        float uscl = p.texWidth;
        float vscl = p.texHeight;

        // d[s,t]/d[X,Y].
        float dsdx = uvDA.x * uscl;
        float dsdy = uvDA.y * uscl;
        float dtdx = uvDA.z * vscl;
        float dtdy = uvDA.w * vscl;

        // Calculate footprint axis lengths.
        float A = dsdx*dsdx + dtdx*dtdx;
        float B = dsdy*dsdy + dtdy*dtdy;
        float C = dsdx*dsdy + dtdx*dtdy;
        float l2b = 0.5 * (A + B);
        float l2n = 0.25 * (A-B)*(A-B) + C*C;
        float l2a = sqrt(l2n);
        float lenMinorSqr = fmaxf(0.0, l2b - l2a);
        float lenMajorSqr = l2b + l2a;

        // Footprint vs. mip level gradient.
        if (pdw && FILTER_MODE == TEX_MODE_LINEAR_MIPMAP_LINEAR)
        {
            float dw   = 0.72134752f / (l2n + l2a * l2b); // Constant is 0.5/ln(2).
            float AB   = dw * .5f * (A - B);
            float Cw   = dw * C;
            float l2aw = dw * l2a;
            float d_f_ddsdX = uscl * (dsdx * (l2aw + AB) + dsdy * Cw);
            float d_f_ddsdY = uscl * (dsdy * (l2aw - AB) + dsdx * Cw);
            float d_f_ddtdX = vscl * (dtdx * (l2aw + AB) + dtdy * Cw);
            float d_f_ddtdY = vscl * (dtdy * (l2aw - AB) + dtdx * Cw);

            *pdw = make_float4(d_f_ddsdX, d_f_ddsdY, d_f_ddtdX, d_f_ddtdY);

            // In cube maps, there is also a texture coordinate vs. mip level gradient.
            if (CUBE_MODE)
            {
                float4 dx, dy, dz;
                indexCubeMapGrad2(uv, dvdX, dvdY, dx, dy, dz);

                float3 d_dsdX_dv = make_float3(dx.x, dy.x, dz.x);
                float3 d_dsdY_dv = make_float3(dx.y, dy.y, dz.y);
                float3 d_dtdX_dv = make_float3(dx.z, dy.z, dz.z);
                float3 d_dtdY_dv = make_float3(dx.w, dy.w, dz.w);

                float3 d_f_dv = make_float3(0.f, 0.f, 0.f);
                d_f_dv += d_dsdX_dv * d_f_ddsdX;
                d_f_dv += d_dsdY_dv * d_f_ddsdY;
                d_f_dv += d_dtdX_dv * d_f_ddtdX;
                d_f_dv += d_dtdY_dv * d_f_ddtdY;

                *pdfdv = d_f_dv;
            }
        }

        // Finally, calculate mip level.
        flevel = .5f * __log2f(lenMajorSqr);
    }

    // Bias the mip level and clamp.
    if (p.mipLevelBias)
        flevel += p.mipLevelBias[pidx];
    flevel = fminf(fmaxf(flevel, 0.f), (float)p.mipLevelMax);

    // Calculate levels depending on filter mode.
    level0 = __float2int_rd(flevel);

    // Leave everything else at zero if flevel == 0 (magnification) or when in linear-mipmap-nearest mode.
    if (FILTER_MODE == TEX_MODE_LINEAR_MIPMAP_LINEAR && flevel > 0.f)
    {
        level1 = min(level0 + 1, p.mipLevelMax);
        flevel -= level0; // Fractional part. Zero if clamped on last level.
    }
}

//------------------------------------------------------------------------
// Texel fetch and accumulator helpers that understand cube map corners.

template<class T>
static __device__ __forceinline__ void fetchQuad(T& a00, T& a10, T& a01, T& a11, const float* pIn, int4 tc, bool corner)
{
    if (corner)
    {
        T avg = zero_value<T>();
        if (tc.x >= 0) avg += (a00 = *((const T*)&pIn[tc.x]));
        if (tc.y >= 0) avg += (a10 = *((const T*)&pIn[tc.y]));
        if (tc.z >= 0) avg += (a01 = *((const T*)&pIn[tc.z]));
        if (tc.w >= 0) avg += (a11 = *((const T*)&pIn[tc.w]));
        avg *= 0.33333333f;
        if (tc.x < 0) a00 = avg;
        if (tc.y < 0) a10 = avg;
        if (tc.z < 0) a01 = avg;
        if (tc.w < 0) a11 = avg;
    }
    else
    {
        a00 = (tc.x >= 0) ? *((const T*)&pIn[tc.x]) : zero_value<T>();
        a10 = (tc.y >= 0) ? *((const T*)&pIn[tc.y]) : zero_value<T>();
        a01 = (tc.z >= 0) ? *((const T*)&pIn[tc.z]) : zero_value<T>();
        a11 = (tc.w >= 0) ? *((const T*)&pIn[tc.w]) : zero_value<T>();
    }
}

static __device__ __forceinline__ void accumQuad(float4 c, float* pOut, int level, int4 tc, bool corner, CA_TEMP_PARAM)
{
    if (corner)
    {
        float cb;
        if (tc.x < 0) cb = c.x;
        if (tc.y < 0) cb = c.y;
        if (tc.z < 0) cb = c.z;
        if (tc.w < 0) cb = c.w;
        cb *= 0.33333333f;
        if (tc.x >= 0) caAtomicAddTexture(pOut, level, tc.x, c.x + cb);
        if (tc.y >= 0) caAtomicAddTexture(pOut, level, tc.y, c.y + cb);
        if (tc.z >= 0) caAtomicAddTexture(pOut, level, tc.z, c.z + cb);
        if (tc.w >= 0) caAtomicAddTexture(pOut, level, tc.w, c.w + cb);
    }
    else
    {
        if (tc.x >= 0) caAtomicAddTexture(pOut, level, tc.x, c.x);
        if (tc.y >= 0) caAtomicAddTexture(pOut, level, tc.y, c.y);
        if (tc.z >= 0) caAtomicAddTexture(pOut, level, tc.z, c.z);
        if (tc.w >= 0) caAtomicAddTexture(pOut, level, tc.w, c.w);
    }
}

//------------------------------------------------------------------------
// Mip builder kernel.

template<class T, int C>
static __forceinline__ __device__ void MipBuildKernelTemplate(const TextureKernelParams p)
{
    // Sizes.
    int2 sz_in = mipLevelSize(p, p.mipLevelOut - 1);
    int2 sz_out = mipLevelSize(p, p.mipLevelOut);

    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= sz_out.x || py >= sz_out.y)
        return;

    // Pixel indices.
    int pidx_in0 = p.channels * (((px + sz_in.x * py) << 1) + (pz * sz_in.x * sz_in.y));
    int pidx_in1 = pidx_in0 + p.channels * sz_in.x; // Next pixel down.
    int pidx_out = p.channels * (px + sz_out.x * (py + sz_out.y * pz));

    // Input and output pointers.
    const float* pin = p.tex[p.mipLevelOut - 1];
    float* pout = (float*)p.tex[p.mipLevelOut];

    // Special case: Input texture height or width is 1.
    if (sz_in.x == 1 || sz_in.y == 1)
    {
        if (sz_in.y == 1)
            pidx_in1 = pidx_in0 + p.channels; // Next pixel on the right.

        for (int i=0; i < p.channels; i += C)
        {
            T v0 = *((const T*)&pin[pidx_in0 + i]);
            T v1 = *((const T*)&pin[pidx_in1 + i]);
            T avg = .5f * (v0 + v1);
#if TEX_DEBUG_MIP_RETAIN_VARIANCE
            avg = (avg - .5f) * 1.41421356f + .5f;
#endif
            *((T*)&pout[pidx_out + i]) = avg;
        }

        return;
    }

    for (int i=0; i < p.channels; i += C)
    {
        T v0 = *((const T*)&pin[pidx_in0 + i]);
        T v1 = *((const T*)&pin[pidx_in0 + i + p.channels]);
        T v2 = *((const T*)&pin[pidx_in1 + i]);
        T v3 = *((const T*)&pin[pidx_in1 + i + p.channels]);
        T avg = .25f * (v0 + v1 + v2 + v3);
#if TEX_DEBUG_MIP_RETAIN_VARIANCE
        avg = (avg - .5f) * 2.f + .5f;
#endif
        *((T*)&pout[pidx_out + i]) = avg;
    }
}

// Template specializations.
__global__ void MipBuildKernel1(const TextureKernelParams p) { MipBuildKernelTemplate<float,  1>(p); }
__global__ void MipBuildKernel2(const TextureKernelParams p) { MipBuildKernelTemplate<float2, 2>(p); }
__global__ void MipBuildKernel4(const TextureKernelParams p) { MipBuildKernelTemplate<float4, 4>(p); }

//------------------------------------------------------------------------
// Forward kernel.

template <class T, int C, bool CUBE_MODE, bool BIAS_ONLY, int FILTER_MODE>
static __forceinline__ __device__ void TextureFwdKernelTemplate(const TextureKernelParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    int tz = (p.texDepth == 1) ? 0 : pz;
    if (px >= p.imgWidth || py >= p.imgHeight || pz >= p.n)
        return;

    // Pixel index.
    int pidx = px + p.imgWidth * (py + p.imgHeight * pz);

    // Output ptr.
    float* pOut = p.out + pidx * p.channels;

    // Get UV.
    float3 uv;
    if (CUBE_MODE)
        uv = ((const float3*)p.uv)[pidx];
    else
        uv = make_float3(((const float2*)p.uv)[pidx], 0.f);

    // Nearest mode.
    if (FILTER_MODE == TEX_MODE_NEAREST)
    {
        int tc = indexTextureNearest<CUBE_MODE>(p, uv, tz);
        tc *= p.channels;
        const float* pIn = p.tex[0];

        // Copy if valid tc, otherwise output zero.
        for (int i=0; i < p.channels; i += C)
            *((T*)&pOut[i]) = (tc >= 0) ? *((const T*)&pIn[tc + i]) : zero_value<T>();

        return; // Exit.
    }

    // Calculate mip level. In 'linear' mode these will all stay zero.
    float  flevel = 0.f; // Fractional level.
    int    level0 = 0;   // Discrete level 0.
    int    level1 = 0;   // Discrete level 1.
    calculateMipLevel<CUBE_MODE, BIAS_ONLY, FILTER_MODE>(level0, level1, flevel, p, pidx, uv, 0, 0);

    // Get texel indices and pointer for level 0.
    int4 tc0 = make_int4(0, 0, 0, 0);
    float2 uv0 = indexTextureLinear<CUBE_MODE>(p, uv, tz, tc0, level0);
    const float* pIn0 = p.tex[level0];
    bool corner0 = CUBE_MODE && ((tc0.x | tc0.y | tc0.z | tc0.w) < 0);
    tc0 *= p.channels;

    // Bilinear fetch.
    if (FILTER_MODE == TEX_MODE_LINEAR || FILTER_MODE == TEX_MODE_LINEAR_MIPMAP_NEAREST)
    {
        // Interpolate.
        for (int i=0; i < p.channels; i += C, tc0 += C)
        {
            T a00, a10, a01, a11;
            fetchQuad<T>(a00, a10, a01, a11, pIn0, tc0, corner0);
            *((T*)&pOut[i]) = bilerp(a00, a10, a01, a11, uv0);
        }
        return; // Exit.
    }

    // Get texel indices and pointer for level 1.
    int4 tc1 = make_int4(0, 0, 0, 0);
    float2 uv1 = indexTextureLinear<CUBE_MODE>(p, uv, tz, tc1, level1);
    const float* pIn1 = p.tex[level1];
    bool corner1 = CUBE_MODE && ((tc1.x | tc1.y | tc1.z | tc1.w) < 0);
    tc1 *= p.channels;

    // Trilinear fetch.
    for (int i=0; i < p.channels; i += C, tc0 += C, tc1 += C)
    {
        // First level.
        T a00, a10, a01, a11;
        fetchQuad<T>(a00, a10, a01, a11, pIn0, tc0, corner0);
        T a = bilerp(a00, a10, a01, a11, uv0);

        // Second level unless in magnification mode.
        if (flevel > 0.f)
        {
            T b00, b10, b01, b11;
            fetchQuad<T>(b00, b10, b01, b11, pIn1, tc1, corner1);
            T b = bilerp(b00, b10, b01, b11, uv1);
            a = lerp(a, b, flevel); // Interpolate between levels.
        }

        // Write.
        *((T*)&pOut[i]) = a;
    }
}

// Template specializations.
__global__ void TextureFwdKernelNearest1                    (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, false, false, TEX_MODE_NEAREST>(p); }
__global__ void TextureFwdKernelNearest2                    (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, false, false, TEX_MODE_NEAREST>(p); }
__global__ void TextureFwdKernelNearest4                    (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, false, false, TEX_MODE_NEAREST>(p); }
__global__ void TextureFwdKernelLinear1                     (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, false, false, TEX_MODE_LINEAR>(p); }
__global__ void TextureFwdKernelLinear2                     (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, false, false, TEX_MODE_LINEAR>(p); }
__global__ void TextureFwdKernelLinear4                     (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, false, false, TEX_MODE_LINEAR>(p); }
__global__ void TextureFwdKernelLinearMipmapNearest1        (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, false, false, TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelLinearMipmapNearest2        (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, false, false, TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelLinearMipmapNearest4        (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, false, false, TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelLinearMipmapLinear1         (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, false, false, TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelLinearMipmapLinear2         (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, false, false, TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelLinearMipmapLinear4         (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, false, false, TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelCubeNearest1                (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, true,  false, TEX_MODE_NEAREST>(p); }
__global__ void TextureFwdKernelCubeNearest2                (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, true,  false, TEX_MODE_NEAREST>(p); }
__global__ void TextureFwdKernelCubeNearest4                (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, true,  false, TEX_MODE_NEAREST>(p); }
__global__ void TextureFwdKernelCubeLinear1                 (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, true,  false, TEX_MODE_LINEAR>(p); }
__global__ void TextureFwdKernelCubeLinear2                 (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, true,  false, TEX_MODE_LINEAR>(p); }
__global__ void TextureFwdKernelCubeLinear4                 (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, true,  false, TEX_MODE_LINEAR>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapNearest1    (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, true,  false, TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapNearest2    (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, true,  false, TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapNearest4    (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, true,  false, TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapLinear1     (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, true,  false, TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapLinear2     (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, true,  false, TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapLinear4     (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, true,  false, TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelLinearMipmapNearestBO1      (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, false, true,  TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelLinearMipmapNearestBO2      (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, false, true,  TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelLinearMipmapNearestBO4      (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, false, true,  TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelLinearMipmapLinearBO1       (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, false, true,  TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelLinearMipmapLinearBO2       (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, false, true,  TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelLinearMipmapLinearBO4       (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, false, true,  TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapNearestBO1  (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, true,  true,  TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapNearestBO2  (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, true,  true,  TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapNearestBO4  (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, true,  true,  TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapLinearBO1   (const TextureKernelParams p) { TextureFwdKernelTemplate<float,  1, true,  true,  TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapLinearBO2   (const TextureKernelParams p) { TextureFwdKernelTemplate<float2, 2, true,  true,  TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureFwdKernelCubeLinearMipmapLinearBO4   (const TextureKernelParams p) { TextureFwdKernelTemplate<float4, 4, true,  true,  TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }

//------------------------------------------------------------------------
// Gradient mip puller kernel.

template<class T, int C>
static __forceinline__ __device__ void MipGradKernelTemplate(const TextureKernelParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.texWidth || py >= p.texHeight)
        return;

    // Number of wide elements.
    int c = p.channels;
    if (C == 2) c >>= 1;
    if (C == 4) c >>= 2;

    // Dynamically allocated shared memory for holding a texel.
    extern __shared__ float s_texelAccum[];
    int sharedOfs = threadIdx.x + threadIdx.y * blockDim.x;
    int sharedStride = blockDim.x * blockDim.y;
#   define TEXEL_ACCUM(_i) (s_texelAccum + (sharedOfs + (_i) * sharedStride))

    // Clear the texel.
    for (int i=0; i < p.channels; i++)
        *TEXEL_ACCUM(i) = 0.f;

    // Track texel position and accumulation weight over the mip stack.
    int x = px;
    int y = py;
    float w = 1.f;

    // Pull gradients from all levels.
    int2 sz = mipLevelSize(p, 0); // Previous level size.
    for (int level=1; level <= p.mipLevelMax; level++)
    {
        // Weight decay depends on previous level size.
        if (sz.x > 1) w *= .5f;
        if (sz.y > 1) w *= .5f;

        // Current level size and coordinates.
        sz = mipLevelSize(p, level);
        x >>= 1;
        y >>= 1;

        T* pIn = (T*)(p.gradTex[level] + (x + sz.x * (y + sz.y * pz)) * p.channels);
        for (int i=0; i < c; i++)
            accum_from_mem(TEXEL_ACCUM(i * C), sharedStride, pIn[i], w);
    }

    // Add to main texture gradients.
    T* pOut = (T*)(p.gradTex[0] + (px + p.texWidth * (py + p.texHeight * pz)) * p.channels);
    for (int i=0; i < c; i++)
        accum_to_mem(pOut[i], TEXEL_ACCUM(i * C), sharedStride);
}

// Template specializations.
__global__ void MipGradKernel1(const TextureKernelParams p) { MipGradKernelTemplate<float,  1>(p); }
__global__ void MipGradKernel2(const TextureKernelParams p) { MipGradKernelTemplate<float2, 2>(p); }
__global__ void MipGradKernel4(const TextureKernelParams p) { MipGradKernelTemplate<float4, 4>(p); }

//------------------------------------------------------------------------
// Gradient kernel.

template <bool CUBE_MODE, bool BIAS_ONLY, int FILTER_MODE>
static __forceinline__ __device__ void TextureGradKernelTemplate(const TextureKernelParams p)
{
    // Temporary space for coalesced atomics.
    CA_DECLARE_TEMP(TEX_GRAD_MAX_KERNEL_BLOCK_WIDTH * TEX_GRAD_MAX_KERNEL_BLOCK_HEIGHT);

    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    int tz = (p.texDepth == 1) ? 0 : pz;
    if (px >= p.imgWidth || py >= p.imgHeight || pz >= p.n)
        return;

    // Pixel index.
    int pidx = px + p.imgWidth * (py + p.imgHeight * pz);

    // Early exit if output gradients are zero.
    const float* pDy = p.dy + pidx * p.channels;
    unsigned int dmax = 0u;
    if ((p.channels & 3) == 0)
    {
        for (int i=0; i < p.channels; i += 4)
        {
            uint4 dy = *((const uint4*)&pDy[i]);
            dmax |= (dy.x | dy.y | dy.z | dy.w);
        }
    }
    else
    {
        for (int i=0; i < p.channels; i++)
            dmax |= __float_as_uint(pDy[i]);
    }

    // Store zeros and exit.
    if (__uint_as_float(dmax) == 0.f)
    {
        if (CUBE_MODE)
        {
            if (FILTER_MODE != TEX_MODE_NEAREST)
                ((float3*)p.gradUV)[pidx] = make_float3(0.f, 0.f, 0.f);
            if (FILTER_MODE == TEX_MODE_LINEAR_MIPMAP_LINEAR)
            {
                if (p.gradUVDA)
                {
                    ((float2*)p.gradUVDA)[3 * pidx + 0] = make_float2(0.f, 0.f);
                    ((float2*)p.gradUVDA)[3 * pidx + 1] = make_float2(0.f, 0.f);
                    ((float2*)p.gradUVDA)[3 * pidx + 2] = make_float2(0.f, 0.f);
                }
                if (p.gradMipLevelBias)
                    p.gradMipLevelBias[pidx] = 0.f;
            }
        }
        else
        {
            if (FILTER_MODE != TEX_MODE_NEAREST)
                ((float2*)p.gradUV)[pidx] = make_float2(0.f, 0.f);
            if (FILTER_MODE == TEX_MODE_LINEAR_MIPMAP_LINEAR)
            {
                if (p.gradUVDA)
                    ((float4*)p.gradUVDA)[pidx] = make_float4(0.f, 0.f, 0.f, 0.f);
                if (p.gradMipLevelBias)
                    p.gradMipLevelBias[pidx] = 0.f;
            }
        }
        return;
    }

    // Get UV.
    float3 uv;
    if (CUBE_MODE)
        uv = ((const float3*)p.uv)[pidx];
    else
        uv = make_float3(((const float2*)p.uv)[pidx], 0.f);

    // Nearest mode - texture gradients only.
    if (FILTER_MODE == TEX_MODE_NEAREST)
    {
        int tc = indexTextureNearest<CUBE_MODE>(p, uv, tz);
        if (tc < 0)
            return; // Outside texture.

        tc *= p.channels;
        float* pOut = p.gradTex[0];

        // Accumulate texture gradients.
        for (int i=0; i < p.channels; i++)
            caAtomicAddTexture(pOut, 0, tc + i, pDy[i]);

        return; // Exit.
    }

    // Calculate mip level. In 'linear' mode these will all stay zero.
    float4 dw = make_float4(0.f, 0.f, 0.f, 0.f);
    float3 dfdv = make_float3(0.f, 0.f, 0.f);
    float  flevel = 0.f; // Fractional level.
    int    level0 = 0;   // Discrete level 0.
    int    level1 = 0;   // Discrete level 1.
    calculateMipLevel<CUBE_MODE, BIAS_ONLY, FILTER_MODE>(level0, level1, flevel, p, pidx, uv, &dw, &dfdv);

    // UV gradient accumulators.
    float gu = 0.f;
    float gv = 0.f;

    // Get texel indices and pointers for level 0.
    int4 tc0 = make_int4(0, 0, 0, 0);
    float2 uv0 = indexTextureLinear<CUBE_MODE>(p, uv, tz, tc0, level0);
    const float* pIn0 = p.tex[level0];
    float* pOut0 = p.gradTex[level0];
    bool corner0 = CUBE_MODE && ((tc0.x | tc0.y | tc0.z | tc0.w) < 0);
    tc0 *= p.channels;

    // Texel weights.
    float uv011 = uv0.x * uv0.y;
    float uv010 = uv0.x - uv011;
    float uv001 = uv0.y - uv011;
    float uv000 = 1.f - uv0.x - uv001;
    float4 tw0 = make_float4(uv000, uv010, uv001, uv011);

    // Attribute weights.
    int2 sz0 = mipLevelSize(p, level0);
    float sclu0 = (float)sz0.x;
    float sclv0 = (float)sz0.y;

    // Bilinear mode - texture and uv gradients.
    if (FILTER_MODE == TEX_MODE_LINEAR || FILTER_MODE == TEX_MODE_LINEAR_MIPMAP_NEAREST)
    {
        for (int i=0; i < p.channels; i++, tc0 += 1)
        {
            float dy = pDy[i];
            accumQuad(tw0 * dy, pOut0, level0, tc0, corner0, CA_TEMP);

            float a00, a10, a01, a11;
            fetchQuad<float>(a00, a10, a01, a11, pIn0, tc0, corner0);
            float ad = (a11 + a00 - a10 - a01);
            gu += dy * ((a10 - a00) + uv0.y * ad) * sclu0;
            gv += dy * ((a01 - a00) + uv0.x * ad) * sclv0;
        }

        // Store UV gradients and exit.
        if (CUBE_MODE)
            ((float3*)p.gradUV)[pidx] = indexCubeMapGrad(uv, gu, gv);
        else
            ((float2*)p.gradUV)[pidx] = make_float2(gu, gv);

        return;
    }

    // Accumulate fractional mip level gradient.
    float df = 0; // dL/df.

    // Get texel indices and pointers for level 1.
    int4 tc1 = make_int4(0, 0, 0, 0);
    float2 uv1 = indexTextureLinear<CUBE_MODE>(p, uv, tz, tc1, level1);
    const float* pIn1 = p.tex[level1];
    float* pOut1 = p.gradTex[level1];
    bool corner1 = CUBE_MODE && ((tc1.x | tc1.y | tc1.z | tc1.w) < 0);
    tc1 *= p.channels;

    // Texel weights.
    float uv111 = uv1.x * uv1.y;
    float uv110 = uv1.x - uv111;
    float uv101 = uv1.y - uv111;
    float uv100 = 1.f - uv1.x - uv101;
    float4 tw1 = make_float4(uv100, uv110, uv101, uv111);

    // Attribute weights.
    int2 sz1 = mipLevelSize(p, level1);
    float sclu1 = (float)sz1.x;
    float sclv1 = (float)sz1.y;

    // Trilinear mode.
    for (int i=0; i < p.channels; i++, tc0 += 1, tc1 += 1)
    {
        float dy = pDy[i];
        float dy0 = (1.f - flevel) * dy;
        accumQuad(tw0 * dy0, pOut0, level0, tc0, corner0, CA_TEMP);

        // UV gradients for first level.
        float a00, a10, a01, a11;
        fetchQuad<float>(a00, a10, a01, a11, pIn0, tc0, corner0);
        float ad = (a11 + a00 - a10 - a01);
        gu += dy0 * ((a10 - a00) + uv0.y * ad) * sclu0;
        gv += dy0 * ((a01 - a00) + uv0.x * ad) * sclv0;

        // Second level unless in magnification mode.
        if (flevel > 0.f)
        {
            // Texture gradients for second level.
            float dy1 = flevel * dy;
            accumQuad(tw1 * dy1, pOut1, level1, tc1, corner1, CA_TEMP);

            // UV gradients for second level.
            float b00, b10, b01, b11;
            fetchQuad<float>(b00, b10, b01, b11, pIn1, tc1, corner1);
            float bd = (b11 + b00 - b10 - b01);
            gu += dy1 * ((b10 - b00) + uv1.y * bd) * sclu1;
            gv += dy1 * ((b01 - b00) + uv1.x * bd) * sclv1;

            // Mip level gradient.
            float a = bilerp(a00, a10, a01, a11, uv0);
            float b = bilerp(b00, b10, b01, b11, uv1);
            df += (b-a) * dy;
        }
    }

    // Store UV gradients.
    if (CUBE_MODE)
        ((float3*)p.gradUV)[pidx] = indexCubeMapGrad(uv, gu, gv) + (dfdv * df);
    else
        ((float2*)p.gradUV)[pidx] = make_float2(gu, gv);

    // Store mip level bias gradient.
    if (p.gradMipLevelBias)
        p.gradMipLevelBias[pidx] = df;

    // Store UV pixel differential gradients.
    if (!BIAS_ONLY)
    {
        // Final gradients.
        dw *= df; // dL/(d{s,y}/d{X,Y}) = df/(d{s,y}/d{X,Y}) * dL/df.

        // Store them.
        if (CUBE_MODE)
        {
            // Remap from dL/(d{s,t}/s{X,Y}) to dL/(d{x,y,z}/d{X,Y}).
            float3 g0, g1;
            indexCubeMapGrad4(uv, dw, g0, g1);
            ((float2*)p.gradUVDA)[3 * pidx + 0] = make_float2(g0.x, g1.x);
            ((float2*)p.gradUVDA)[3 * pidx + 1] = make_float2(g0.y, g1.y);
            ((float2*)p.gradUVDA)[3 * pidx + 2] = make_float2(g0.z, g1.z);
        }
        else
            ((float4*)p.gradUVDA)[pidx] = dw;
    }
}

// Template specializations.
__global__ void TextureGradKernelNearest                    (const TextureKernelParams p) { TextureGradKernelTemplate<false, false, TEX_MODE_NEAREST>(p); }
__global__ void TextureGradKernelLinear                     (const TextureKernelParams p) { TextureGradKernelTemplate<false, false, TEX_MODE_LINEAR>(p); }
__global__ void TextureGradKernelLinearMipmapNearest        (const TextureKernelParams p) { TextureGradKernelTemplate<false, false, TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureGradKernelLinearMipmapLinear         (const TextureKernelParams p) { TextureGradKernelTemplate<false, false, TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureGradKernelCubeNearest                (const TextureKernelParams p) { TextureGradKernelTemplate<true,  false, TEX_MODE_NEAREST>(p); }
__global__ void TextureGradKernelCubeLinear                 (const TextureKernelParams p) { TextureGradKernelTemplate<true,  false, TEX_MODE_LINEAR>(p); }
__global__ void TextureGradKernelCubeLinearMipmapNearest    (const TextureKernelParams p) { TextureGradKernelTemplate<true,  false, TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureGradKernelCubeLinearMipmapLinear     (const TextureKernelParams p) { TextureGradKernelTemplate<true,  false, TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureGradKernelLinearMipmapNearestBO      (const TextureKernelParams p) { TextureGradKernelTemplate<false, true,  TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureGradKernelLinearMipmapLinearBO       (const TextureKernelParams p) { TextureGradKernelTemplate<false, true,  TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }
__global__ void TextureGradKernelCubeLinearMipmapNearestBO  (const TextureKernelParams p) { TextureGradKernelTemplate<true,  true,  TEX_MODE_LINEAR_MIPMAP_NEAREST>(p); }
__global__ void TextureGradKernelCubeLinearMipmapLinearBO   (const TextureKernelParams p) { TextureGradKernelTemplate<true,  true,  TEX_MODE_LINEAR_MIPMAP_LINEAR>(p); }

//------------------------------------------------------------------------
