#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include "common.h"
#include "rasterize.h"

//------------------------------------------------------------------------
// Gradient Cuda kernel.

template <bool ENABLE_DB>
static __forceinline__ __device__ void RasterizeGradKernelTemplate(const RasterizeGradParams p)
{
    // Temporary space for coalesced atomics.
    CA_DECLARE_TEMP(RAST_GRAD_MAX_KERNEL_BLOCK_WIDTH * RAST_GRAD_MAX_KERNEL_BLOCK_HEIGHT);    

    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.width || py >= p.height || pz >= p.depth)
        return;

    // Pixel index.
    int pidx = px + p.width * (py + p.height * pz);

    // Read triangle idx and dy.
    float2 dy  = ((float2*)p.dy)[pidx * 2];
    float4 ddb = ENABLE_DB ? ((float4*)p.ddb)[pidx] : make_float4(0.f, 0.f, 0.f, 0.f);
    int triIdx = (int)(((float*)p.out)[pidx * 4 + 3]) - 1;

    // Exit if nothing to do.
    if (triIdx < 0 || triIdx >= p.numTriangles)
        return; // No or corrupt triangle.
    int grad_all_dy = __float_as_int(dy.x) | __float_as_int(dy.y); // Bitwise OR of all incoming gradients.
    int grad_all_ddb = 0;
    if (ENABLE_DB)
        grad_all_ddb = __float_as_int(ddb.x) | __float_as_int(ddb.y) | __float_as_int(ddb.z) | __float_as_int(ddb.w);
    if (((grad_all_dy | grad_all_ddb) << 1) == 0)
        return; // All incoming gradients are +0/-0.

    // Fetch vertex indices.
    int vi0 = p.tri[triIdx * 3 + 0];
    int vi1 = p.tri[triIdx * 3 + 1];
    int vi2 = p.tri[triIdx * 3 + 2];

    // Bail out if vertex indices are corrupt.
    if (vi0 < 0 || vi0 >= p.numVertices ||
        vi1 < 0 || vi1 >= p.numVertices ||
        vi2 < 0 || vi2 >= p.numVertices)
        return;

    // In instance mode, adjust vertex indices by minibatch index.
    if (p.instance_mode)
    {
        vi0 += pz * p.numVertices;
        vi1 += pz * p.numVertices;
        vi2 += pz * p.numVertices;
    }

    // Initialize coalesced atomics.
    CA_SET_GROUP(triIdx);
    
    // Fetch vertex positions.
    float4 p0 = ((float4*)p.pos)[vi0];
    float4 p1 = ((float4*)p.pos)[vi1];
    float4 p2 = ((float4*)p.pos)[vi2];

    // Evaluate edge functions.
    float fx = p.xs * (float)px + p.xo;
    float fy = p.ys * (float)py + p.yo;
    float p0x = p0.x - fx * p0.w;
    float p0y = p0.y - fy * p0.w;
    float p1x = p1.x - fx * p1.w;
    float p1y = p1.y - fy * p1.w;
    float p2x = p2.x - fx * p2.w;
    float p2y = p2.y - fy * p2.w;
    float a0 = p1x*p2y - p1y*p2x;
    float a1 = p2x*p0y - p2y*p0x;
    float a2 = p0x*p1y - p0y*p1x;

    // Compute inverse area with epsilon.
    float at = a0 + a1 + a2;
    float ep = copysignf(1e-6f, at); // ~1 pixel in 1k x 1k image.
    float iw = 1.f / (at + ep);

    // Perspective correct, normalized barycentrics.
    float b0 = a0 * iw;
    float b1 = a1 * iw;

    // Position gradients.
    float gb0  = dy.x * iw;
    float gb1  = dy.y * iw;
    float gbb  = gb0 * b0 + gb1 * b1;
    float gp0x = gbb * (p2y - p1y) - gb1 * p2y;
    float gp1x = gbb * (p0y - p2y) + gb0 * p2y;
    float gp2x = gbb * (p1y - p0y) - gb0 * p1y + gb1 * p0y;
    float gp0y = gbb * (p1x - p2x) + gb1 * p2x;
    float gp1y = gbb * (p2x - p0x) - gb0 * p2x;
    float gp2y = gbb * (p0x - p1x) + gb0 * p1x - gb1 * p0x;
    float gp0w = -fx * gp0x - fy * gp0y;
    float gp1w = -fx * gp1x - fy * gp1y;
    float gp2w = -fx * gp2x - fy * gp2y;

    // Bary differential gradients.
    if (ENABLE_DB && ((grad_all_ddb) << 1) != 0)
    {
        float dfxdX = p.xs * iw;
        float dfydY = p.ys * iw;
        ddb.x *= dfxdX;
        ddb.y *= dfydY;
        ddb.z *= dfxdX;
        ddb.w *= dfydY;

        float da0dX = p1.y * p2.w - p2.y * p1.w;
        float da1dX = p2.y * p0.w - p0.y * p2.w;
        float da2dX = p0.y * p1.w - p1.y * p0.w;
        float da0dY = p2.x * p1.w - p1.x * p2.w;
        float da1dY = p0.x * p2.w - p2.x * p0.w;
        float da2dY = p1.x * p0.w - p0.x * p1.w;
        float datdX = da0dX + da1dX + da2dX;
        float datdY = da0dY + da1dY + da2dY;

        float x01 = p0.x - p1.x;
        float x12 = p1.x - p2.x;
        float x20 = p2.x - p0.x;
        float y01 = p0.y - p1.y;
        float y12 = p1.y - p2.y;
        float y20 = p2.y - p0.y;
        float w01 = p0.w - p1.w;
        float w12 = p1.w - p2.w;
        float w20 = p2.w - p0.w;

        float a0p1 = fy * p2.x - fx * p2.y;
        float a0p2 = fx * p1.y - fy * p1.x;
        float a1p0 = fx * p2.y - fy * p2.x;
        float a1p2 = fy * p0.x - fx * p0.y;

        float wdudX = 2.f * b0 * datdX - da0dX; 
        float wdudY = 2.f * b0 * datdY - da0dY;
        float wdvdX = 2.f * b1 * datdX - da1dX;
        float wdvdY = 2.f * b1 * datdY - da1dY;

        float c0  = iw * (ddb.x * wdudX + ddb.y * wdudY + ddb.z * wdvdX + ddb.w * wdvdY);
        float cx  = c0 * fx - ddb.x * b0 - ddb.z * b1;
        float cy  = c0 * fy - ddb.y * b0 - ddb.w * b1;
        float cxy = iw * (ddb.x * datdX + ddb.y * datdY);
        float czw = iw * (ddb.z * datdX + ddb.w * datdY);

        gp0x += c0 * y12 - cy * w12              + czw * p2y                                               + ddb.w * p2.w;
        gp1x += c0 * y20 - cy * w20 - cxy * p2y                              - ddb.y * p2.w;
        gp2x += c0 * y01 - cy * w01 + cxy * p1y  - czw * p0y                 + ddb.y * p1.w                - ddb.w * p0.w;
        gp0y += cx * w12 - c0 * x12              - czw * p2x                                - ddb.z * p2.w;
        gp1y += cx * w20 - c0 * x20 + cxy * p2x               + ddb.x * p2.w;
        gp2y += cx * w01 - c0 * x01 - cxy * p1x  + czw * p0x  - ddb.x * p1.w                + ddb.z * p0.w;
        gp0w += cy * x12 - cx * y12              - czw * a1p0                               + ddb.z * p2.y - ddb.w * p2.x;
        gp1w += cy * x20 - cx * y20 - cxy * a0p1              - ddb.x * p2.y + ddb.y * p2.x;
        gp2w += cy * x01 - cx * y01 - cxy * a0p2 - czw * a1p2 + ddb.x * p1.y - ddb.y * p1.x - ddb.z * p0.y + ddb.w * p0.x;
    }

    // Accumulate using coalesced atomics.
    caAtomicAdd3_xyw(p.grad + 4 * vi0, gp0x, gp0y, gp0w);
    caAtomicAdd3_xyw(p.grad + 4 * vi1, gp1x, gp1y, gp1w);
    caAtomicAdd3_xyw(p.grad + 4 * vi2, gp2x, gp2y, gp2w);
}

// Template specializations.
__global__ void RasterizeGradKernel  (const RasterizeGradParams p) { RasterizeGradKernelTemplate<false>(p); }
__global__ void RasterizeGradKernelDb(const RasterizeGradParams p) { RasterizeGradKernelTemplate<true>(p); }

//------------------------------------------------------------------------
