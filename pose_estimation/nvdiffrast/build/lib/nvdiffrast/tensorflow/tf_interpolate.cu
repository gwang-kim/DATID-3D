#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

//------------------------------------------------------------------------
// Common op attribute parser.

static __host__ void interpolateParseOpAttributes(OpKernelConstruction* ctx, InterpolateKernelParams& p, bool enableDA)
{
    if (enableDA)
    {
        OP_REQUIRES_OK(ctx, ctx->GetAttr("diff_attrs_all", &p.diff_attrs_all));
        if (!p.diff_attrs_all)
        {
            std::vector<int> diff_attrs_vec;
            OP_REQUIRES_OK(ctx, ctx->GetAttr("diff_attrs", &diff_attrs_vec));
            OP_REQUIRES(ctx, diff_attrs_vec.size() > 0, errors::InvalidArgument("differentiation enabled with empty diff_attrs list"));
            OP_REQUIRES(ctx, diff_attrs_vec.size() <= IP_MAX_DIFF_ATTRS, errors::InvalidArgument("too many entries in diff_attrs list (increase IP_MAX_DIFF_ATTRS)"));
            p.numDiffAttr = diff_attrs_vec.size();
            memcpy(p.diffAttrs, &diff_attrs_vec[0], diff_attrs_vec.size()*sizeof(int));
        }
    }
}

//------------------------------------------------------------------------
// Forward TensorFlow op.

template <bool ENABLE_DA>
struct InterpolateFwdOp : public OpKernel
{
    InterpolateKernelParams m_attribs;

    InterpolateFwdOp(OpKernelConstruction* ctx): OpKernel(ctx)
    {
        memset(&m_attribs, 0, sizeof(m_attribs));
        interpolateParseOpAttributes(ctx, m_attribs, ENABLE_DA);
    }

    void Compute(OpKernelContext* ctx)
    {
        InterpolateKernelParams& p = m_attribs;
        hipStream_t stream = ctx->eigen_device<Eigen::GpuDevice>().stream();

        // Get input.
        const Tensor& attr    = ctx->input(0);
        const Tensor& rast    = ctx->input(1);
        const Tensor& tri     = ctx->input(2);
        const Tensor& rast_db = ctx->input(ENABLE_DA ? 3 : 2);

        // Instance rendering mode?
        p.instance_mode = attr.dims() > 2;

        // Extract input dimensions.
        if (p.instance_mode)
        {
            p.numVertices  = (attr.dims() > 1) ? attr.dim_size(1) : 0;
            p.numAttr      = (attr.dims() > 2) ? attr.dim_size(2) : 0;
        }
        else
        {
            p.numVertices  = (attr.dims() > 0) ? attr.dim_size(0) : 0;
            p.numAttr      = (attr.dims() > 1) ? attr.dim_size(1) : 0;
        }
        p.numTriangles = (tri.dims() > 0) ? tri.dim_size(0) : 0;
        p.height       = (rast.dims() > 1) ? rast.dim_size(1) : 0;
        p.width        = (rast.dims() > 2) ? rast.dim_size(2) : 0;
        p.depth        = (rast.dims() > 0) ? rast.dim_size(0) : 0;

        // Sanity checks.
        OP_REQUIRES(ctx, rast.dims() == 4 && rast.dim_size(0) > 0 && rast.dim_size(1) > 0 && rast.dim_size(2) > 0 && rast.dim_size(3) == 4, errors::InvalidArgument("rast must have shape[>0, >0, >0, 4]"));
        OP_REQUIRES(ctx, tri.dims() == 2 && tri.dim_size(0) > 0 && tri.dim_size(1) == 3, errors::InvalidArgument("tri must have shape [>0, 3]"));
        OP_REQUIRES(ctx, (attr.dims() == 2 || attr.dims() == 3) && attr.dim_size(0) > 0 && attr.dim_size(1) > 0 && (attr.dims() == 2 || attr.dim_size(2) > 0), errors::InvalidArgument("attr must have shape [>0, >0, >0] or [>0, >0]"));
        if (p.instance_mode)
            OP_REQUIRES(ctx, attr.dim_size(0) == p.depth || attr.dim_size(0) == 1, errors::InvalidArgument("minibatch size mismatch between inputs rast, attr"));
        if (ENABLE_DA)
        {
            OP_REQUIRES(ctx, rast_db.dims() == 4 && rast_db.dim_size(0) > 0 && rast_db.dim_size(1) > 0 && rast_db.dim_size(2) > 0 && rast_db.dim_size(3) == 4, errors::InvalidArgument("rast_db must have shape[>0, >0, >0, 4]"));
            OP_REQUIRES(ctx, rast_db.dim_size(1) == rast.dim_size(1) && rast_db.dim_size(2) == rast.dim_size(2), errors::InvalidArgument("spatial size mismatch between inputs rast and rast_db"));
            OP_REQUIRES(ctx, rast_db.dim_size(0) == p.depth, errors::InvalidArgument("minibatch size mismatch between inputs rast, rast_db"));
        }

        // All diff attrs mode.
        if (p.diff_attrs_all)
            p.numDiffAttr = p.numAttr;

        // Get input pointers.
        p.attr = attr.flat<float>().data();
        p.rast = rast.flat<float>().data();
        p.tri = tri.flat<int>().data();
        p.attrBC = (p.instance_mode && attr.dim_size(0) == 1) ? 1 : 0;
        p.rastDB = ENABLE_DA ? rast_db.flat<float>().data() : 0;

        // Allocate main output tensor.
        Tensor* out_tensor = NULL;
        TensorShape out_shape;
        out_shape.AddDim(p.depth);
        out_shape.AddDim(p.height);
        out_shape.AddDim(p.width);
        out_shape.AddDim(p.numAttr);
        OP_REQUIRES_OK(ctx, ctx->allocate_output(0, out_shape, &out_tensor));
        p.out = out_tensor->flat<float>().data();

        // Allocate pixel differential output tensor.
        Tensor* out_da_tensor = NULL;
        out_shape.set_dim(3, p.numDiffAttr * 2);
        OP_REQUIRES_OK(ctx, ctx->allocate_output(1, out_shape, &out_da_tensor));
        p.outDA = ENABLE_DA ? out_da_tensor->flat<float>().data() : 0;

        // Verify that buffers are aligned to allow float2/float4 operations.
        OP_REQUIRES(ctx, !((uintptr_t)p.rast   & 15), errors::Internal("rast input tensor not aligned to float4"));
        OP_REQUIRES(ctx, !((uintptr_t)p.rastDB & 15), errors::Internal("rast_db input tensor not aligned to float4"));        
        if (ENABLE_DA)
            OP_REQUIRES(ctx, !((uintptr_t)p.outDA & 7), errors::Internal("out_da output tensor not aligned to float2"));

        // Choose launch parameters.
        dim3 blockSize = getLaunchBlockSize(IP_FWD_MAX_KERNEL_BLOCK_WIDTH, IP_FWD_MAX_KERNEL_BLOCK_HEIGHT, p.width, p.height);
        dim3 gridSize  = getLaunchGridSize(blockSize, p.width, p.height, p.depth);

        // Launch CUDA kernel.
        void* args[] = {&p};
        void* func = ENABLE_DA ? (void*)InterpolateFwdKernelDa : (void*)InterpolateFwdKernel;
        OP_CHECK_CUDA_ERROR(ctx, hipLaunchKernel(reinterpret_cast<const void*>(func), gridSize, blockSize, args, 0, stream));
    }
};

REGISTER_OP("InterpolateFwd")
    .Input      ("attr: float")
    .Input      ("rast: float")
    .Input      ("tri: int32")
    .Output     ("out: float")
    .Output     ("out_da: float");

REGISTER_OP("InterpolateFwdDa")
    .Input      ("attr: float")
    .Input      ("rast: float")
    .Input      ("tri: int32")
    .Input      ("rast_db: float")
    .Output     ("out: float")
    .Output     ("out_da: float")
    .Attr       ("diff_attrs_all: int")
    .Attr       ("diff_attrs: list(int)");

REGISTER_KERNEL_BUILDER(Name("InterpolateFwd")  .Device(DEVICE_GPU), InterpolateFwdOp<false>);
REGISTER_KERNEL_BUILDER(Name("InterpolateFwdDa").Device(DEVICE_GPU), InterpolateFwdOp<true>);

//------------------------------------------------------------------------
// Gradient TensorFlow op.

template <bool ENABLE_DA>
struct InterpolateGradOp : public OpKernel
{
    InterpolateKernelParams m_attribs;

    InterpolateGradOp(OpKernelConstruction* ctx): OpKernel(ctx)
    {
        memset(&m_attribs, 0, sizeof(m_attribs));
        interpolateParseOpAttributes(ctx, m_attribs, ENABLE_DA);      
    }

    void Compute(OpKernelContext* ctx)
    {
        InterpolateKernelParams& p = m_attribs;
        hipStream_t stream = ctx->eigen_device<Eigen::GpuDevice>().stream();

        // Get input.
        const Tensor& attr    = ctx->input(0);
        const Tensor& rast    = ctx->input(1);
        const Tensor& tri     = ctx->input(2);
        const Tensor& dy      = ctx->input(3);
        const Tensor& rast_db = ctx->input(ENABLE_DA ? 4 : 3);
        const Tensor& dda     = ctx->input(ENABLE_DA ? 5 : 3);

        // Instance rendering mode?
        p.instance_mode = attr.dims() > 2;

        // Extract input dimensions.
        if (p.instance_mode)
        {
            p.numVertices  = (attr.dims() > 1) ? attr.dim_size(1) : 0;
            p.numAttr      = (attr.dims() > 2) ? attr.dim_size(2) : 0;
        }
        else
        {
            p.numVertices  = (attr.dims() > 0) ? attr.dim_size(0) : 0;
            p.numAttr      = (attr.dims() > 1) ? attr.dim_size(1) : 0;
        }
        p.numTriangles = (tri.dims() > 0) ? tri.dim_size(0) : 0;
        p.depth        = (rast.dims() > 0) ? rast.dim_size(0) : 0;
        p.height       = (rast.dims() > 1) ? rast.dim_size(1) : 0;
        p.width        = (rast.dims() > 2) ? rast.dim_size(2) : 0;
        int attr_depth = p.instance_mode ? (attr.dims() > 1 ? attr.dim_size(0) : 0) : 1;

        // Sanity checks.
        OP_REQUIRES(ctx, rast.dims() == 4 && rast.dim_size(0) > 0 && rast.dim_size(1) > 0 && rast.dim_size(2) > 0 && rast.dim_size(3) == 4, errors::InvalidArgument("rast must have shape[>0, >0, >0, 4]"));
        OP_REQUIRES(ctx, tri.dims() == 2 && tri.dim_size(0) > 0 && tri.dim_size(1) == 3, errors::InvalidArgument("tri must have shape [>0, 3]"));
        OP_REQUIRES(ctx, (attr.dims() == 2 || attr.dims() == 3) && attr.dim_size(0) > 0 && attr.dim_size(1) > 0 && (attr.dims() == 2 || attr.dim_size(2) > 0), errors::InvalidArgument("attr must have shape [>0, >0, >0] or [>0, >0]"));
        OP_REQUIRES(ctx, dy.dims() == 4 && dy.dim_size(0) > 0 && dy.dim_size(1) == p.height && dy.dim_size(2) == p.width && dy.dim_size(3) > 0, errors::InvalidArgument("dy must have shape [>0, height, width, >0]"));
        OP_REQUIRES(ctx, dy.dim_size(3) == p.numAttr, errors::InvalidArgument("argument count mismatch between inputs dy, attr"));
        OP_REQUIRES(ctx, (attr_depth == p.depth || attr_depth == 1) && dy.dim_size(0) == p.depth, errors::InvalidArgument("minibatch size mismatch between inputs rast, dy, attr"));
        if (ENABLE_DA)
        {
            OP_REQUIRES(ctx, dda.dims() == 4 && dda.dim_size(0) > 0 && dda.dim_size(1) == p.height && dda.dim_size(2) == p.width, errors::InvalidArgument("dda must have shape [>0, height, width, ?]"));
            OP_REQUIRES(ctx, dda.dim_size(0) == p.depth, errors::InvalidArgument("minibatch size mismatch between rast, dda"));
        }

        // All diff attrs mode.
        if (p.diff_attrs_all)
            p.numDiffAttr = p.numAttr;

        // Get input pointers.
        p.attr   = attr.flat<float>().data();
        p.rast   = rast.flat<float>().data();
        p.tri    = tri.flat<int>().data();
        p.dy     = dy.flat<float>().data();
        p.rastDB = ENABLE_DA ? rast_db.flat<float>().data() : 0;
        p.dda    = ENABLE_DA ? dda.flat<float>().data() : 0;
        p.attrBC = (p.instance_mode && attr_depth < p.depth) ? 1 : 0;

        // Allocate attribute gradient output tensor.
        Tensor* grad_attr_tensor = NULL;
        TensorShape grad_attr_shape;
        if (p.instance_mode)
            grad_attr_shape.AddDim(attr_depth);
        grad_attr_shape.AddDim(p.numVertices);
        grad_attr_shape.AddDim(p.numAttr);
        OP_REQUIRES_OK(ctx, ctx->allocate_output(0, grad_attr_shape, &grad_attr_tensor));
        p.gradAttr = grad_attr_tensor->flat<float>().data();

        // Allocate bary gradient output tensor.
        Tensor* grad_rast_tensor = NULL;
        TensorShape grad_rast_shape;
        grad_rast_shape.AddDim(p.depth);
        grad_rast_shape.AddDim(p.height);
        grad_rast_shape.AddDim(p.width);
        grad_rast_shape.AddDim(4);
        OP_REQUIRES_OK(ctx, ctx->allocate_output(1, grad_rast_shape, &grad_rast_tensor));
        p.gradRaster = grad_rast_tensor->flat<float>().data();

        // Allocate bary pixel diff gradient output tensor.
        if (ENABLE_DA)
        {
            Tensor* grad_rast_db_tensor = NULL;
            OP_REQUIRES_OK(ctx, ctx->allocate_output(2, grad_rast_shape, &grad_rast_db_tensor));
            p.gradRasterDB = grad_rast_db_tensor->flat<float>().data();
        }
        
        // Clear attribute gradients.
        hipMemsetAsync(p.gradAttr, 0, attr_depth * p.numVertices * p.numAttr * sizeof(float), stream);

        // Verify that buffers are aligned to allow float2/float4 operations.
        OP_REQUIRES(ctx, !((uintptr_t)p.rast   & 15), errors::Internal("rast input tensor not aligned to float4"));
        OP_REQUIRES(ctx, !((uintptr_t)p.gradRaster & 15), errors::Internal("grad_rast output tensor not aligned to float4"));
        if (ENABLE_DA)
        {
            OP_REQUIRES(ctx, !((uintptr_t)p.dda & 7), errors::Internal("dda input tensor not aligned to float2"));
            OP_REQUIRES(ctx, !((uintptr_t)p.rastDB & 15), errors::Internal("rast_db input tensor not aligned to float4"));        
            OP_REQUIRES(ctx, !((uintptr_t)p.gradRasterDB & 15), errors::Internal("grad_rast_db output tensor not aligned to float4"));
        }
    
        // Choose launch parameters.
        dim3 blockSize = getLaunchBlockSize(IP_GRAD_MAX_KERNEL_BLOCK_WIDTH, IP_GRAD_MAX_KERNEL_BLOCK_HEIGHT, p.width, p.height);
        dim3 gridSize  = getLaunchGridSize(blockSize, p.width, p.height, p.depth);

        // Launch CUDA kernel.
        void* args[] = {&p};
        void* func = ENABLE_DA ? (void*)InterpolateGradKernelDa : (void*)InterpolateGradKernel;
        OP_CHECK_CUDA_ERROR(ctx, hipLaunchKernel(reinterpret_cast<const void*>(func), gridSize, blockSize, args, 0, stream));
    }
};

REGISTER_OP("InterpolateGrad")
    .Input      ("attr: float")
    .Input      ("rast: float")
    .Input      ("tri: int32")
    .Input      ("dy: float")
    .Output     ("grad_attr: float")
    .Output     ("grad_rast: float")
    ;

REGISTER_OP("InterpolateGradDa")
    .Input      ("attr: float")
    .Input      ("rast: float")
    .Input      ("tri: int32")
    .Input      ("dy: float")
    .Input      ("rast_db: float")
    .Input      ("dda: float")
    .Output     ("grad_attr: float")
    .Output     ("grad_rast: float")
    .Output     ("grad_rast_db: float")
    .Attr       ("diff_attrs_all: int")
    .Attr       ("diff_attrs: list(int)");
    ;

REGISTER_KERNEL_BUILDER(Name("InterpolateGrad")  .Device(DEVICE_GPU), InterpolateGradOp<false>);
REGISTER_KERNEL_BUILDER(Name("InterpolateGradDa").Device(DEVICE_GPU), InterpolateGradOp<true>);

//------------------------------------------------------------------------
