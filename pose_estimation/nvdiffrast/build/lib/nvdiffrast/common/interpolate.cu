#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include "common.h"
#include "interpolate.h"

//------------------------------------------------------------------------
// Forward kernel.

template <bool ENABLE_DA>
static __forceinline__ __device__ void InterpolateFwdKernelTemplate(const InterpolateKernelParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.width || py >= p.height || pz >= p.depth)
        return;

    // Pixel index.
    int pidx = px + p.width * (py + p.height * pz);

    // Output ptrs.
    float* out = p.out + pidx * p.numAttr;
    float2* outDA = ENABLE_DA ? (((float2*)p.outDA) + pidx * p.numDiffAttr) : 0;

    // Fetch rasterizer output.
    float4 r = ((float4*)p.rast)[pidx];
    int triIdx = (int)r.w - 1;
    bool triValid = (triIdx >= 0 && triIdx < p.numTriangles);

    // If no geometry in entire warp, zero the output and exit.
    // Otherwise force barys to zero and output with live threads.
    if (__all_sync(0xffffffffu, !triValid))
    {
        for (int i=0; i < p.numAttr; i++)
            out[i] = 0.f;
        if (ENABLE_DA)
            for (int i=0; i < p.numDiffAttr; i++)
                outDA[i] = make_float2(0.f, 0.f);
        return;
    }

    // Fetch vertex indices.
    int vi0 = triValid ? p.tri[triIdx * 3 + 0] : 0;
    int vi1 = triValid ? p.tri[triIdx * 3 + 1] : 0;
    int vi2 = triValid ? p.tri[triIdx * 3 + 2] : 0;

    // Bail out if corrupt indices.
    if (vi0 < 0 || vi0 >= p.numVertices ||
        vi1 < 0 || vi1 >= p.numVertices ||
        vi2 < 0 || vi2 >= p.numVertices)
        return;

    // In instance mode, adjust vertex indices by minibatch index unless broadcasting.
    if (p.instance_mode && !p.attrBC)
    {
        vi0 += pz * p.numVertices;
        vi1 += pz * p.numVertices;
        vi2 += pz * p.numVertices;
    }

    // Pointers to attributes.
    const float* a0 = p.attr + vi0 * p.numAttr;
    const float* a1 = p.attr + vi1 * p.numAttr;
    const float* a2 = p.attr + vi2 * p.numAttr;

    // Barys. If no triangle, force all to zero -> output is zero.
    float b0 = triValid ? r.x : 0.f;
    float b1 = triValid ? r.y : 0.f;
    float b2 = triValid ? (1.f - r.x - r.y) : 0.f;

    // Interpolate and write attributes.
    for (int i=0; i < p.numAttr; i++)
        out[i] = b0*a0[i] + b1*a1[i] + b2*a2[i];

    // No diff attrs? Exit.
    if (!ENABLE_DA)
        return;

    // Read bary pixel differentials if we have a triangle.
    float4 db = make_float4(0.f, 0.f, 0.f, 0.f);
    if (triValid)
        db = ((float4*)p.rastDB)[pidx];

    // Unpack a bit.
    float dudx = db.x;
    float dudy = db.y;
    float dvdx = db.z;
    float dvdy = db.w;

    // Calculate the pixel differentials of chosen attributes.    
    for (int i=0; i < p.numDiffAttr; i++)
    {   
        // Input attribute index.
        int j = p.diff_attrs_all ? i : p.diffAttrs[i];
        if (j < 0)
            j += p.numAttr; // Python-style negative indices.

        // Zero output if invalid index.
        float dsdx = 0.f;
        float dsdy = 0.f;
        if (j >= 0 && j < p.numAttr)
        {
            float s0 = a0[j];
            float s1 = a1[j];
            float s2 = a2[j];
            float dsdu = s0 - s2;
            float dsdv = s1 - s2;
            dsdx = dudx*dsdu + dvdx*dsdv;
            dsdy = dudy*dsdu + dvdy*dsdv;
        }

        // Write.
        outDA[i] = make_float2(dsdx, dsdy);
    }
}

// Template specializations.
__global__ void InterpolateFwdKernel  (const InterpolateKernelParams p) { InterpolateFwdKernelTemplate<false>(p); }
__global__ void InterpolateFwdKernelDa(const InterpolateKernelParams p) { InterpolateFwdKernelTemplate<true>(p); }

//------------------------------------------------------------------------
// Gradient kernel.

template <bool ENABLE_DA>
static __forceinline__ __device__ void InterpolateGradKernelTemplate(const InterpolateKernelParams p)
{
    // Temporary space for coalesced atomics.
    CA_DECLARE_TEMP(IP_GRAD_MAX_KERNEL_BLOCK_WIDTH * IP_GRAD_MAX_KERNEL_BLOCK_HEIGHT);    

    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.width || py >= p.height || pz >= p.depth)
        return;

    // Pixel index.
    int pidx = px + p.width * (py + p.height * pz);

    // Fetch triangle ID. If none, output zero bary/db gradients and exit.
    float4 r = ((float4*)p.rast)[pidx];
    int triIdx = (int)r.w - 1;
    if (triIdx < 0 || triIdx >= p.numTriangles)
    {
        ((float4*)p.gradRaster)[pidx] = make_float4(0.f, 0.f, 0.f, 0.f);
        if (ENABLE_DA)
            ((float4*)p.gradRasterDB)[pidx] = make_float4(0.f, 0.f, 0.f, 0.f);
        return;
    }

    // Fetch vertex indices.
    int vi0 = p.tri[triIdx * 3 + 0];
    int vi1 = p.tri[triIdx * 3 + 1];
    int vi2 = p.tri[triIdx * 3 + 2];

    // Bail out if corrupt indices.
    if (vi0 < 0 || vi0 >= p.numVertices ||
        vi1 < 0 || vi1 >= p.numVertices ||
        vi2 < 0 || vi2 >= p.numVertices)
        return;

    // In instance mode, adjust vertex indices by minibatch index unless broadcasting.
    if (p.instance_mode && !p.attrBC)
    {
        vi0 += pz * p.numVertices;
        vi1 += pz * p.numVertices;
        vi2 += pz * p.numVertices;
    }

    // Initialize coalesced atomics.
    CA_SET_GROUP(triIdx);

    // Pointers to inputs.
    const float* a0 = p.attr + vi0 * p.numAttr;
    const float* a1 = p.attr + vi1 * p.numAttr;
    const float* a2 = p.attr + vi2 * p.numAttr;
    const float* pdy = p.dy + pidx * p.numAttr;

    // Pointers to outputs.
    float* ga0 = p.gradAttr + vi0 * p.numAttr;
    float* ga1 = p.gradAttr + vi1 * p.numAttr;
    float* ga2 = p.gradAttr + vi2 * p.numAttr;

    // Barys and bary gradient accumulators.
    float b0 = r.x;
    float b1 = r.y;
    float b2 = 1.f - r.x - r.y;
    float gb0 = 0.f;
    float gb1 = 0.f;

    // Loop over attributes and accumulate attribute gradients.
    for (int i=0; i < p.numAttr; i++)
    {
        float y = pdy[i];
        float s0 = a0[i];
        float s1 = a1[i];
        float s2 = a2[i];
        gb0 += y * (s0 - s2);
        gb1 += y * (s1 - s2);
        caAtomicAdd(ga0 + i, b0 * y);
        caAtomicAdd(ga1 + i, b1 * y);
        caAtomicAdd(ga2 + i, b2 * y);
    }

    // Write the bary gradients.
    ((float4*)p.gradRaster)[pidx] = make_float4(gb0, gb1, 0.f, 0.f);

    // If pixel differentials disabled, we're done.
    if (!ENABLE_DA)
        return;

    // Calculate gradients based on attribute pixel differentials.
    const float2* dda = ((float2*)p.dda) + pidx * p.numDiffAttr;
    float gdudx = 0.f;
    float gdudy = 0.f;
    float gdvdx = 0.f;
    float gdvdy = 0.f;

    // Read bary pixel differentials.
    float4 db = ((float4*)p.rastDB)[pidx];
    float dudx = db.x;
    float dudy = db.y;
    float dvdx = db.z;
    float dvdy = db.w;

    for (int i=0; i < p.numDiffAttr; i++)
    {
        // Input attribute index.
        int j = p.diff_attrs_all ? i : p.diffAttrs[i];
        if (j < 0)
            j += p.numAttr; // Python-style negative indices.

        // Check that index is valid.
        if (j >= 0 && j < p.numAttr)
        {
            float2 dsdxy = dda[i];
            float dsdx = dsdxy.x;
            float dsdy = dsdxy.y;

            float s0 = a0[j];
            float s1 = a1[j];
            float s2 = a2[j];

            // Gradients of db.
            float dsdu = s0 - s2;
            float dsdv = s1 - s2;
            gdudx += dsdu * dsdx;
            gdudy += dsdu * dsdy;
            gdvdx += dsdv * dsdx;
            gdvdy += dsdv * dsdy;

            // Gradients of attributes.
            float du = dsdx*dudx + dsdy*dudy;
            float dv = dsdx*dvdx + dsdy*dvdy;
            caAtomicAdd(ga0 + j, du);
            caAtomicAdd(ga1 + j, dv);
            caAtomicAdd(ga2 + j, -du - dv);
        }
    }

    // Write.
    ((float4*)p.gradRasterDB)[pidx] = make_float4(gdudx, gdudy, gdvdx, gdvdy);
}

// Template specializations.
__global__ void InterpolateGradKernel  (const InterpolateKernelParams p) { InterpolateGradKernelTemplate<false>(p); }
__global__ void InterpolateGradKernelDa(const InterpolateKernelParams p) { InterpolateGradKernelTemplate<true>(p); }

//------------------------------------------------------------------------
