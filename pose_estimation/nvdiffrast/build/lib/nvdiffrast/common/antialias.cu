#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include "antialias.h"

//------------------------------------------------------------------------
// Helpers.

#define F32_MAX (3.402823466e+38f)
static __forceinline__ __device__ bool same_sign(float a, float b) { return (__float_as_int(a) ^ __float_as_int(b)) >= 0; }
static __forceinline__ __device__ bool rational_gt(float n0, float n1, float d0, float d1) { return (n0*d1 > n1*d0) == same_sign(d0, d1); }
static __forceinline__ __device__ int max_idx3(float n0, float n1, float n2, float d0, float d1, float d2)
{
    bool g10 = rational_gt(n1, n0, d1, d0);
    bool g20 = rational_gt(n2, n0, d2, d0);
    bool g21 = rational_gt(n2, n1, d2, d1);
    if (g20 && g21) return 2;
    if (g10) return 1;
    return 0;
}

//------------------------------------------------------------------------
// Format of antialiasing work items stored in work buffer. Usually accessed directly as int4.

struct AAWorkItem
{
    enum
    {
        EDGE_MASK       = 3,    // Edge index in lowest bits.
        FLAG_DOWN_BIT   = 2,    // Down instead of right.
        FLAG_TRI1_BIT   = 3,    // Edge is from other pixel's triangle.
    };

    int             px, py;         // Pixel x, y.
    unsigned int    pz_flags;       // High 16 bits = pixel z, low 16 bits = edge index and flags.
    float           alpha;          // Antialiasing alpha value. Zero if no AA.
};

//------------------------------------------------------------------------
// Hash functions. Adapted from public-domain code at http://www.burtleburtle.net/bob/hash/doobs.html

#define JENKINS_MAGIC (0x9e3779b9u)
static __device__ __forceinline__ void jenkins_mix(unsigned int& a, unsigned int& b, unsigned int& c)
{
    a -= b; a -= c; a ^= (c>>13);
    b -= c; b -= a; b ^= (a<<8);
    c -= a; c -= b; c ^= (b>>13);
    a -= b; a -= c; a ^= (c>>12);
    b -= c; b -= a; b ^= (a<<16);
    c -= a; c -= b; c ^= (b>>5);
    a -= b; a -= c; a ^= (c>>3);
    b -= c; b -= a; b ^= (a<<10);
    c -= a; c -= b; c ^= (b>>15);
}

// Helper class for hash index iteration. Implements simple odd-skip linear probing with a key-dependent skip.
class HashIndex
{
public:
    __device__ __forceinline__ HashIndex(const AntialiasKernelParams& p, uint64_t key)
    {
        m_mask = p.allocTriangles * AA_HASH_ELEMENTS_PER_TRIANGLE - 1;
        m_idx  = (uint32_t)(key & 0xffffffffu);
        m_skip = (uint32_t)(key >> 32);
        uint32_t dummy = JENKINS_MAGIC;
        jenkins_mix(m_idx, m_skip, dummy);
        m_idx &= m_mask;
        m_skip &= m_mask;
        m_skip |= 1;
    }
    __device__ __forceinline__ int get(void) const { return m_idx; }
    __device__ __forceinline__ void next(void) { m_idx = (m_idx + m_skip) & m_mask; }
private:
    uint32_t m_idx, m_skip, m_mask;
};

static __device__ __forceinline__ void hash_insert(const AntialiasKernelParams& p, uint64_t key, int v)
{
    HashIndex idx(p, key);
    while(1)
    {
        uint64_t prev = atomicCAS((unsigned long long*)&p.evHash[idx.get()], 0, (unsigned long long)key);
        if (prev == 0 || prev == key)
            break;
        idx.next();
    }
    int* q = (int*)&p.evHash[idx.get()];
    int a = atomicCAS(q+2, 0, v);
    if (a != 0 && a != v)
        atomicCAS(q+3, 0, v);
}

static __device__ __forceinline__ int2 hash_find(const AntialiasKernelParams& p, uint64_t key)
{
    HashIndex idx(p, key);
    while(1)
    {
        uint4 entry = p.evHash[idx.get()];
        uint64_t k = ((uint64_t)entry.x) | (((uint64_t)entry.y) << 32);
        if (k == key || k == 0)
            return make_int2((int)entry.z, (int)entry.w);
        idx.next();
    }
}

static __device__ __forceinline__ void evhash_insert_vertex(const AntialiasKernelParams& p, int va, int vb, int vn)
{
    if (va == vb)
        return;
    
    uint64_t v0 = (uint32_t)min(va, vb) + 1; // canonical vertex order
    uint64_t v1 = (uint32_t)max(va, vb) + 1;
    uint64_t vk = v0 | (v1 << 32); // hash key
    hash_insert(p, vk, vn + 1);
}

static __forceinline__ __device__ int evhash_find_vertex(const AntialiasKernelParams& p, int va, int vb, int vr)
{
    if (va == vb)
        return -1;

    uint64_t v0 = (uint32_t)min(va, vb) + 1; // canonical vertex order
    uint64_t v1 = (uint32_t)max(va, vb) + 1;
    uint64_t vk = v0 | (v1 << 32); // hash key
    int2 vn = hash_find(p, vk) - 1;
    if (vn.x == vr) return vn.y;
    if (vn.y == vr) return vn.x;
    return -1;
}

//------------------------------------------------------------------------
// Mesh analysis kernel.

__global__ void AntialiasFwdMeshKernel(const AntialiasKernelParams p)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= p.numTriangles)
        return;

    int v0 = p.tri[idx * 3 + 0];
    int v1 = p.tri[idx * 3 + 1];
    int v2 = p.tri[idx * 3 + 2];

    if (v0 < 0 || v0 >= p.numVertices ||
        v1 < 0 || v1 >= p.numVertices ||
        v2 < 0 || v2 >= p.numVertices)
        return;

    if (v0 == v1 || v1 == v2 || v2 == v0)
        return;

    evhash_insert_vertex(p, v1, v2, v0);
    evhash_insert_vertex(p, v2, v0, v1);
    evhash_insert_vertex(p, v0, v1, v2);
}

//------------------------------------------------------------------------
// Discontinuity finder kernel.

__global__ void AntialiasFwdDiscontinuityKernel(const AntialiasKernelParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * AA_DISCONTINUITY_KERNEL_BLOCK_WIDTH + threadIdx.x;
    int py = blockIdx.y * AA_DISCONTINUITY_KERNEL_BLOCK_HEIGHT + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.width || py >= p.height || pz >= p.n)
        return;

    // Pointer to our TriIdx and fetch.
    int pidx0 = ((px + p.width * (py + p.height * pz)) << 2) + 3;
    float tri0 = p.rasterOut[pidx0];

    // Look right, clamp at edge.
    int pidx1 = pidx0;
    if (px < p.width - 1)
        pidx1 += 4;
    float tri1 = p.rasterOut[pidx1];

    // Look down, clamp at edge.
    int pidx2 = pidx0;
    if (py < p.height - 1)
        pidx2 += p.width << 2;
    float tri2 = p.rasterOut[pidx2];

    // Determine amount of work.
    int count = 0;
    if (tri1 != tri0) count  = 1;
    if (tri2 != tri0) count += 1;
    if (!count)
        return; // Exit warp.

    // Coalesce work counter update to once per CTA.
    __shared__ int s_temp;
    s_temp = 0;
    __syncthreads();
    int idx = atomicAdd(&s_temp, count);
    __syncthreads();
    if (idx == 0)
    {
        int base = atomicAdd(&p.workBuffer[0].x, s_temp);
        s_temp = base + 1; // don't clobber the counters in first slot.
    }
    __syncthreads();
    idx += s_temp;

    // Write to memory.
    if (tri1 != tri0) p.workBuffer[idx++] = make_int4(px, py, (pz << 16), 0);
    if (tri2 != tri0) p.workBuffer[idx]   = make_int4(px, py, (pz << 16) + (1 << AAWorkItem::FLAG_DOWN_BIT), 0);
}

//------------------------------------------------------------------------
// Forward analysis kernel.

__global__ void AntialiasFwdAnalysisKernel(const AntialiasKernelParams p)
{
    __shared__ int s_base;
    int workCount = p.workBuffer[0].x;
    for(;;)
    {
        // Persistent threads work fetcher.
        __syncthreads();
        if (threadIdx.x == 0)
            s_base = atomicAdd(&p.workBuffer[0].y, AA_ANALYSIS_KERNEL_THREADS_PER_BLOCK);
        __syncthreads();
        int thread_idx = s_base + threadIdx.x;
        if (thread_idx >= workCount)
            return;

        int4* pItem = p.workBuffer + thread_idx + 1;
        int4 item = *pItem;
        int px = item.x;
        int py = item.y;
        int pz = (int)(((unsigned int)item.z) >> 16);
        int d  = (item.z >> AAWorkItem::FLAG_DOWN_BIT) & 1;

        int pixel0 = px + p.width * (py + p.height * pz);
        int pixel1 = pixel0 + (d ? p.width : 1);
        float2 zt0 = ((float2*)p.rasterOut)[(pixel0 << 1) + 1];
        float2 zt1 = ((float2*)p.rasterOut)[(pixel1 << 1) + 1];
        int tri0 = (int)zt0.y - 1;
        int tri1 = (int)zt1.y - 1;

        // Select triangle based on background / depth.
        int tri = (tri0 >= 0) ? tri0 : tri1;
        if (tri0 >= 0 && tri1 >= 0)
            tri = (zt0.x < zt1.x) ? tri0 : tri1;
        if (tri == tri1)
        {
            // Calculate with respect to neighbor pixel if chose that triangle.
            px += 1 - d;
            py += d;
        }

        // Bail out if triangle index is corrupt.
        if (tri < 0 || tri >= p.numTriangles)
            continue;

        // Fetch vertex indices.
        int vi0 = p.tri[tri * 3 + 0];
        int vi1 = p.tri[tri * 3 + 1];
        int vi2 = p.tri[tri * 3 + 2];

        // Bail out if vertex indices are corrupt.
        if (vi0 < 0 || vi0 >= p.numVertices ||
            vi1 < 0 || vi1 >= p.numVertices ||
            vi2 < 0 || vi2 >= p.numVertices)
            continue;

        // Fetch opposite vertex indices. Use vertex itself (always silhouette) if no opposite vertex exists.
        int op0 = evhash_find_vertex(p, vi2, vi1, vi0);
        int op1 = evhash_find_vertex(p, vi0, vi2, vi1);
        int op2 = evhash_find_vertex(p, vi1, vi0, vi2);

        // Instance mode: Adjust vertex indices based on minibatch index.
        if (p.instance_mode)
        {
            int vbase = pz * p.numVertices;
            vi0 += vbase; 
            vi1 += vbase; 
            vi2 += vbase;
            if (op0 >= 0) op0 += vbase;
            if (op1 >= 0) op1 += vbase;
            if (op2 >= 0) op2 += vbase;
        }

        // Fetch vertex positions.
        float4 p0 = ((float4*)p.pos)[vi0];
        float4 p1 = ((float4*)p.pos)[vi1];
        float4 p2 = ((float4*)p.pos)[vi2];
        float4 o0 = (op0 < 0) ? p0 : ((float4*)p.pos)[op0];
        float4 o1 = (op1 < 0) ? p1 : ((float4*)p.pos)[op1];
        float4 o2 = (op2 < 0) ? p2 : ((float4*)p.pos)[op2];

        // Project vertices to pixel space.
        float w0  = 1.f / p0.w;
        float w1  = 1.f / p1.w;
        float w2  = 1.f / p2.w;
        float ow0 = 1.f / o0.w;
        float ow1 = 1.f / o1.w;
        float ow2 = 1.f / o2.w;
        float fx  = (float)px + .5f - p.xh;
        float fy  = (float)py + .5f - p.yh;
        float x0  = p0.x * w0 * p.xh - fx;
        float y0  = p0.y * w0 * p.yh - fy;
        float x1  = p1.x * w1 * p.xh - fx;
        float y1  = p1.y * w1 * p.yh - fy;
        float x2  = p2.x * w2 * p.xh - fx;
        float y2  = p2.y * w2 * p.yh - fy;
        float ox0 = o0.x * ow0 * p.xh - fx;
        float oy0 = o0.y * ow0 * p.yh - fy;
        float ox1 = o1.x * ow1 * p.xh - fx;
        float oy1 = o1.y * ow1 * p.yh - fy;
        float ox2 = o2.x * ow2 * p.xh - fx;
        float oy2 = o2.y * ow2 * p.yh - fy;

        // Signs to kill non-silhouette edges.
        float bb = (x1-x0)*(y2-y0) - (x2-x0)*(y1-y0); // Triangle itself.
        float a0 = (x1-ox0)*(y2-oy0) - (x2-ox0)*(y1-oy0); // Wings.
        float a1 = (x2-ox1)*(y0-oy1) - (x0-ox1)*(y2-oy1);
        float a2 = (x0-ox2)*(y1-oy2) - (x1-ox2)*(y0-oy2);

        // If no matching signs anywhere, skip the rest.
        if (same_sign(a0, bb) || same_sign(a1, bb) || same_sign(a2, bb))
        {
            // XY flip for horizontal edges.
            if (d)
            {
                swap(x0, y0);
                swap(x1, y1);
                swap(x2, y2);
            }

            float dx0 = x2 - x1;
            float dx1 = x0 - x2;
            float dx2 = x1 - x0;
            float dy0 = y2 - y1;
            float dy1 = y0 - y2;
            float dy2 = y1 - y0;

            // Check if an edge crosses between us and the neighbor pixel.
            float dc = -F32_MAX;
            float ds = (tri == tri0) ? 1.f : -1.f;
            float d0 = ds * (x1*dy0 - y1*dx0);
            float d1 = ds * (x2*dy1 - y2*dx1);
            float d2 = ds * (x0*dy2 - y0*dx2);

            if (same_sign(y1, y2)) d0 = -F32_MAX, dy0 = 1.f;
            if (same_sign(y2, y0)) d1 = -F32_MAX, dy1 = 1.f;
            if (same_sign(y0, y1)) d2 = -F32_MAX, dy2 = 1.f;

            int di = max_idx3(d0, d1, d2, dy0, dy1, dy2);
            if (di == 0 && same_sign(a0, bb) && fabsf(dy0) >= fabsf(dx0)) dc = d0 / dy0;
            if (di == 1 && same_sign(a1, bb) && fabsf(dy1) >= fabsf(dx1)) dc = d1 / dy1;
            if (di == 2 && same_sign(a2, bb) && fabsf(dy2) >= fabsf(dx2)) dc = d2 / dy2;
            float eps = .0625f; // Expect no more than 1/16 pixel inaccuracy.

            // Adjust output image if a suitable edge was found.
            if (dc > -eps && dc < 1.f + eps)
            {
                dc = fminf(fmaxf(dc, 0.f), 1.f);
                float alpha = ds * (.5f - dc);
                const float* pColor0 = p.color + pixel0 * p.channels;
                const float* pColor1 = p.color + pixel1 * p.channels;
                float* pOutput = p.output + (alpha > 0.f ? pixel0 : pixel1) * p.channels;
                for (int i=0; i < p.channels; i++)
                    atomicAdd(&pOutput[i], alpha * (pColor1[i] - pColor0[i]));

                // Rewrite the work item's flags and alpha. Keep original px, py.
                unsigned int flags = pz << 16;
                flags |= di;
                flags |= d << AAWorkItem::FLAG_DOWN_BIT;
                flags |= (__float_as_uint(ds) >> 31) << AAWorkItem::FLAG_TRI1_BIT;
                ((int2*)pItem)[1] = make_int2(flags, __float_as_int(alpha));
            }
        }
    }
}

//------------------------------------------------------------------------
// Gradient kernel.

__global__ void AntialiasGradKernel(const AntialiasKernelParams p)
{
    // Temporary space for coalesced atomics.
    CA_DECLARE_TEMP(AA_GRAD_KERNEL_THREADS_PER_BLOCK);
    __shared__ int s_base; // Work counter communication across entire CTA.

    int workCount = p.workBuffer[0].x;

    for(;;)
    {
        // Persistent threads work fetcher.
        __syncthreads();
        if (threadIdx.x == 0)
            s_base = atomicAdd(&p.workBuffer[0].y, AA_GRAD_KERNEL_THREADS_PER_BLOCK);
        __syncthreads();
        int thread_idx = s_base + threadIdx.x;
        if (thread_idx >= workCount)
            return;

        // Read work item filled out by forward kernel.
        int4 item = p.workBuffer[thread_idx + 1];
        unsigned int amask = __ballot_sync(0xffffffffu, item.w);
        if (item.w == 0)
            continue; // No effect.

        // Unpack work item and replicate setup from forward analysis kernel.
        int px = item.x;
        int py = item.y;
        int pz = (int)(((unsigned int)item.z) >> 16);
        int d = (item.z >> AAWorkItem::FLAG_DOWN_BIT) & 1;
        float alpha = __int_as_float(item.w);
        int tri1 = (item.z >> AAWorkItem::FLAG_TRI1_BIT) & 1;
        int di = item.z & AAWorkItem::EDGE_MASK;
        float ds = __int_as_float(__float_as_int(1.0) | (tri1 << 31));
        int pixel0 = px + p.width * (py + p.height * pz);
        int pixel1 = pixel0 + (d ? p.width : 1);
        int tri = (int)p.rasterOut[((tri1 ? pixel1 : pixel0) << 2) + 3] - 1;
        if (tri1)
        {
            px += 1 - d;
            py += d;
        }

        // Bail out if triangle index is corrupt.
        bool triFail = (tri < 0 || tri >= p.numTriangles);
        amask = __ballot_sync(amask, !triFail);
        if (triFail)
            continue;

        // Outgoing color gradients.
        float* pGrad0 = p.gradColor + pixel0 * p.channels;
        float* pGrad1 = p.gradColor + pixel1 * p.channels;

        // Incoming color gradients.
        const float* pDy = p.dy + (alpha > 0.f ? pixel0 : pixel1) * p.channels;

        // Position gradient weight based on colors and incoming gradients.
        float dd = 0.f;
        const float* pColor0 = p.color + pixel0 * p.channels;
        const float* pColor1 = p.color + pixel1 * p.channels;

        // Loop over channels and accumulate.
        for (int i=0; i < p.channels; i++)
        {
            float dy = pDy[i];
            if (dy != 0.f)
            {
                // Update position gradient weight.
                dd += dy * (pColor1[i] - pColor0[i]);

                // Update color gradients. No coalescing because all have different targets.
                float v = alpha * dy;
                atomicAdd(&pGrad0[i], -v);
                atomicAdd(&pGrad1[i], v);
            }
        }

        // If position weight is zero, skip the rest.
        bool noGrad = (dd == 0.f);
        amask = __ballot_sync(amask, !noGrad);
        if (noGrad)
            continue;

        // Fetch vertex indices of the active edge and their positions.
        int i1 = (di < 2) ? (di + 1) : 0;
        int i2 = (i1 < 2) ? (i1 + 1) : 0;
        int vi1 = p.tri[3 * tri + i1];
        int vi2 = p.tri[3 * tri + i2];

        // Bail out if vertex indices are corrupt.
        bool vtxFail = (vi1 < 0 || vi1 >= p.numVertices || vi2 < 0 || vi2 >= p.numVertices);
        amask = __ballot_sync(amask, !vtxFail);
        if (vtxFail)
            continue;
    
        // Instance mode: Adjust vertex indices based on minibatch index.
        if (p.instance_mode)
        {
            vi1 += pz * p.numVertices;
            vi2 += pz * p.numVertices;
        }

        // Fetch vertex positions.
        float4 p1 = ((float4*)p.pos)[vi1];
        float4 p2 = ((float4*)p.pos)[vi2];

        // Project vertices to pixel space.
        float pxh = p.xh;
        float pyh = p.yh;
        float fx = (float)px + .5f - pxh;
        float fy = (float)py + .5f - pyh;

        // XY flip for horizontal edges.
        if (d)
        {
            swap(p1.x, p1.y);
            swap(p2.x, p2.y);
            swap(pxh, pyh);
            swap(fx, fy);
        }

        // Gradient calculation setup.
        float w1 = 1.f / p1.w;
        float w2 = 1.f / p2.w;
        float x1 = p1.x * w1 * pxh - fx;
        float y1 = p1.y * w1 * pyh - fy;
        float x2 = p2.x * w2 * pxh - fx;
        float y2 = p2.y * w2 * pyh - fy;
        float dx = x2 - x1;
        float dy = y2 - y1;
        float db = x1*dy - y1*dx;

        // Compute inverse delta-y with epsilon.
        float ep = copysignf(1e-3f, dy); // ~1/1000 pixel.
        float iy = 1.f / (dy + ep);

        // Compute position gradients.
        float dby = db * iy;
        float iw1 = -w1 * iy * dd;
        float iw2 =  w2 * iy * dd;
        float gp1x = iw1 * pxh * y2;
        float gp2x = iw2 * pxh * y1;
        float gp1y = iw1 * pyh * (dby - x2);
        float gp2y = iw2 * pyh * (dby - x1);
        float gp1w = -(p1.x * gp1x + p1.y * gp1y) * w1;
        float gp2w = -(p2.x * gp2x + p2.y * gp2y) * w2;

        // XY flip the gradients.
        if (d)
        {
            swap(gp1x, gp1y);
            swap(gp2x, gp2y);
        }

        // Kill position gradients if alpha was saturated.
        if (fabsf(alpha) >= 0.5f)
        {
            gp1x = gp1y = gp1w = 0.f;
            gp2x = gp2y = gp2w = 0.f;
        }

        // Initialize coalesced atomics. Match both triangle ID and edge index.
        // Also note that some threads may be inactive.
        CA_SET_GROUP_MASK(tri ^ (di << 30), amask);

        // Accumulate gradients.
        caAtomicAdd3_xyw(p.gradPos + 4 * vi1, gp1x, gp1y, gp1w);
        caAtomicAdd3_xyw(p.gradPos + 4 * vi2, gp2x, gp2y, gp2w);
    }
}

//------------------------------------------------------------------------
